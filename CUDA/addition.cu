#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "matrix.cuh"


// CUDA kernel to add two matrices
__global__ void matrixAdditionSimple(int* M1, int* M2, int* M3, int M3R, int M3C) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < M3R * M3C) {
        M3[index] = M1[index] + M2[index];
    }
}

// CUDA kernel to add two matrices in parallel
__global__ void matrixAddition(int* M1, int* M2, int* M3, int M3R, int M3C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M3R && col < M3C) {
        int index = row * M3R + col;
        M3[index] = M1[index] + M2[index];
    }
}

int main() {
    // Variables to measure time spent on a process
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float setupTime;
    float hostToDeviceTime;
    float deviceToHostTime;
    float calculationTime;
    //float shutdownTime;

    // Start the setup timer
    hipEventRecord(start, 0);

    // Define variables
    Matrix M1;
    Matrix M2;
    Matrix M3;
    int M1Rows = 5;
    int M1Cols = 5;
    int M2Rows = 5;
    int M2Cols = 5;
    int M3Rows = M1Rows;
    int M3Cols = M1Cols;

    // Create the matrix objects
    M1 = createMatrix(M1Rows, M1Cols);
    M2 = createMatrix(M2Rows, M2Cols);
    M3 = createMatrix(M3Rows, M3Cols);

    // Populate the matrices
    //populateWithOnes(M1);
    //populateWithOnes(M2);
    populateWithRandomInts(M1);
    populateWithRandomInts(M2);


    // Stop the setup timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&setupTime, start, stop);
    printf("Time spent on setup:                      %f seconds\n", setupTime);

    // Start the data transfer timer (CPU -> GPU / Host -> Device)
    hipEventRecord(start, 0);

    // Allocate memory for matrices on the GPU
    int* device_M1, * device_M2, * device_M3;

    // Allocate memory for matrices on the GPU
    hipMalloc((void**)&device_M1, M1Rows * M1Cols * sizeof(int));
    hipMalloc((void**)&device_M2, M2Rows * M2Cols * sizeof(int));
    hipMalloc((void**)&device_M3, M3Rows * M3Cols * sizeof(int));

    // Copy input matrices from host to device
    hipMemcpy(device_M1, M1.data, M1Rows * M1Cols * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_M2, M2.data, M2Rows * M2Cols * sizeof(int), hipMemcpyHostToDevice);

    // Stop the data transfer timer (CPU -> GPU / Host -> Device)
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&hostToDeviceTime, start, stop);
    printf("Time spent on data transfer (CPU -> GPU): %f seconds\n", hostToDeviceTime);

    // Define block and grid dimensions for CUDA kernel
    dim3 blockDim(32, 32);
    dim3 gridDim((M3Cols + blockDim.x - 1) / blockDim.x, (M3Rows + blockDim.y - 1) / blockDim.y);

    // Start the matrix addition timer
    hipEventRecord(start, 0);

    // Launch the CUDA kernel to perform matrix addition
    matrixAdditionSimple <<<gridDim, blockDim>>> (device_M1, device_M2, device_M3, M3Rows, M3Cols);

    // Stop the matrix addition timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&calculationTime, start, stop);
    printf("Time spent on matrix addition (GPU):      %f seconds\n", calculationTime);

    // Start the data transfer timer (GPU -> CPU / Device -> Host)
    hipEventRecord(start, 0);

    // Copy the result matrix from device to host
    hipMemcpy(M3.data, device_M3, M3Rows * M3Cols * sizeof(int), hipMemcpyDeviceToHost);

    // Stop the  data transfer timer (GPU -> CPU / Device -> Host)
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&deviceToHostTime, start, stop);
    printf("Time spent on data transfer (GPU -> CPU): %f seconds\n", deviceToHostTime);

    // Open a new file to write the result into
    FILE* outputFile = fopen("result.txt", "w");
    if (outputFile == NULL) {
        perror("Unable to create the output file");
        return 1;
    }

    // Write host_M3 to the result file
    for (int i = 0; i < M3Rows; i++) {
        for (int j = 0; j < M3Cols; j++) {
            fprintf(outputFile, "%d ", M3.data[i * M3Rows + j]);
        }
        fprintf(outputFile, "\n");
    }

    // Close the result file
    fclose(outputFile);

    // Deallocate memory on the GPU and CPU
    hipFree(device_M1);
    hipFree(device_M2);
    hipFree(device_M3);

    // End program
    return 0;
}
