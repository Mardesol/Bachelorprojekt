#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>



int main() {
    // Variables to track time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // 
    float setupTime;

    // Start measuring time OS spends on process
    
    hipEventRecord(start, 0);

    int N = 1000;                // Length of rows and cols
    int pattern[] = { 1, 2, 3 };
    
    // Allocate memory for matrices on the CPU
    int* host_M1 = (int*)malloc(N * N * sizeof(int));
    int* host_M2 = (int*)malloc(N * N * sizeof(int));
    int* host_M3 = (int*)malloc(N * N * sizeof(int));

    // Initialize matrices M1 and M2 with the pattern
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            host_M1[i * N + j] = pattern[j % 3];
            host_M2[i * N + j] = pattern[j % 3];
        }
    }

    // End measuring time OS spends on process
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&setupTime, start, stop);

    printf("Time spent on setup: %f seconds\n", setupTime);





    // Allocate memory for matrices on the GPU
    int* device_M1, * device_M2, * device_M3;
    hipMalloc((void**)&device_M1, N * N * sizeof(int));
    hipMalloc((void**)&device_M2, N * N * sizeof(int));
    hipMalloc((void**)&device_M3, N * N * sizeof(int));

    // Copy input matrices from host to device
    hipMemcpy(device_M1, host_M1, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_M2, host_M2, N * N * sizeof(int), hipMemcpyHostToDevice);

    // Define block and grid dimensions for CUDA kernel
    dim3 blockDim(256);  // Use a single thread block
    dim3 gridDim((N * N + blockDim.x - 1) / blockDim.x);


    // Start measuring time for matrix addition on the GPU
    float elapsedTime2;






    hipEventRecord(start, 0);

    // Launch the CUDA kernel to perform matrix addition sequentially on GPU
    matrixAdditionSimple <<<gridDim, blockDim>>> (device_M1, device_M2, device_M3, N);

    // End measuring time for matrix addition on the GPU
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime2, start, stop);

    printf("Time spent on addition (GPU): %f seconds\n", elapsedTime2);

    // Copy the result matrix from device to host
    hipMemcpy(host_M3, device_M3, N * N * sizeof(int), hipMemcpyDeviceToHost);

    // Open a new file to write the result into
    FILE* outputFile = fopen("result.txt", "w");
    if (outputFile == NULL) {
        perror("Unable to create the output file");
        return 1;
    }

    // Write host_M3 to the result file
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            fprintf(outputFile, "%d ", host_M3[i * N + j]);
        }
        fprintf(outputFile, "\n");
    }

    // Close the result file
    fclose(outputFile);

    // Deallocate memory on the GPU and CPU
    hipFree(device_M1);
    hipFree(device_M2);
    hipFree(device_M3);
    free(host_M1);
    free(host_M2);
    free(host_M3);

    // End program
    return 0;
}
