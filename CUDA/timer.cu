#include "timer.cuh"

struct Timer createTimer()
{
	struct Timer timer;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	timer.start = start;
	hipEventCreate(&stop);
	timer.stop = stop;
	return timer;
}

void beginTimer(Timer timer)
{	
	hipEventRecord(timer.start, 0);
}

void endTimer(Timer timer, char* message)
{
	float timeElapsed;
	hipEventRecord(timer.stop, 0);
	hipEventSynchronize(timer.stop);
	hipEventElapsedTime(&timeElapsed, timer.start, timer.stop);
	printf("Time spent on %s: %f seconds\n", message, timeElapsed);
}