#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// CUDA kernel to add two matrices in parallel
__global__ void matrixAddition(int* M1, int* M2, int* M3, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        int index = row * N + col;
        M3[index] = M1[index] + M2[index];
    }
}

int main() {
    // Start measuring time OS spends on process
    clock_t setupBegin = clock();

    int N = 1000;                // Length of rows and cols
    int pattern[] = { 1, 2, 3 };

    // Allocate memory for matrices on the CPU
    int* host_M1 = (int*)malloc(N * N * sizeof(int));
    int* host_M2 = (int*)malloc(N * N * sizeof(int));
    int* host_M3 = (int*)malloc(N * N * sizeof(int));

    // Initialize matrices M1 and M2 with the pattern
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            host_M1[i * N + j] = pattern[j % 3];
            host_M2[i * N + j] = pattern[j % 3];
        }
    }

    // End measuring time OS spends on process
    clock_t setupEnd = clock();
    double time_spent1 = (double)(setupEnd - setupBegin) / CLOCKS_PER_SEC;
    printf("Time spent on setup: %f seconds\n", time_spent1);

    // Allocate memory for matrices on the GPU
    int* device_M1, * device_M2, * device_M3;
    hipMalloc((void**)&device_M1, N * N * sizeof(int));
    hipMalloc((void**)&device_M2, N * N * sizeof(int));
    hipMalloc((void**)&device_M3, N * N * sizeof(int));

    // Copy input matrices from host to device
    hipMemcpy(device_M1, host_M1, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_M2, host_M2, N * N * sizeof(int), hipMemcpyHostToDevice);

    // Define block and grid dimensions for CUDA kernel
    dim3 blockDim(32, 32);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

    // Start measuring time for matrix addition on the GPU
    clock_t begin = clock();

    // Launch the CUDA kernel to perform matrix addition
    matrixAddition <<<gridDim, blockDim>>> (device_M1, device_M2, device_M3, N);

    // End measuring time for matrix addition on the GPU
    clock_t end = clock();
    double time_spent2 = (double)(end - begin) / CLOCKS_PER_SEC;
    printf("Time spent on addition (GPU): %f seconds\n", time_spent2);

    // Copy the result matrix from device to host
    hipMemcpy(host_M3, device_M3, N * N * sizeof(int), hipMemcpyDeviceToHost);

    // Open a new file to write the result into
    FILE* outputFile = fopen("result.txt", "w");
    if (outputFile == NULL) {
        perror("Unable to create the output file");
        return 1;
    }

    // Write host_M3 to the result file
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            fprintf(outputFile, "%d ", host_M3[i * N + j]);
        }
        fprintf(outputFile, "\n");
    }

    // Close the result file
    fclose(outputFile);

    // Deallocate memory on the GPU and CPU
    hipFree(device_M1);
    hipFree(device_M2);
    hipFree(device_M3);
    free(host_M1);
    free(host_M2);
    free(host_M3);

    // End program
    return 0;
}
