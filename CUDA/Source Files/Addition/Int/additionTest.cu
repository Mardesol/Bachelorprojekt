#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include "..\..\Timer\timer.cu" 
#include "..\..\Matrix\matrixInts.cu"

const int rows = 100;
const int cols = 100;

const int M1Rows = rows;
const int M2Rows = rows;
const int M3Rows = rows;

const int M3Cols = cols;
const int M1Cols = cols;
const int M2Cols = cols;

// CUDA kernel to add two matrices sequentially
__global__ void matrixAdditionSequential(int* M1, int* M2, int* M3) {
    for (int i = 0; i < M1Rows; i++) {
        for (int j = 0; j < M1Cols; j++) {
            M3[M1Rows * i + j] = M1[M1Rows * i + j] + M2[M1Rows * i + j];
        }
    }
}

// CUDA kernel to add two matrices in parallel, utilizing thread level parallelism
__global__ void matrixAdditionParallelV1(int* M1, int* M2, int* M3) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < M3Rows * M3Cols) {
        M3[index] = M1[index] + M2[index];
    }
}

// CUDA kernel to add two matrices in parallel, utilizing both thread and block level parallelism
__global__ void matrixAdditionParallelV2(int* M1, int* M2, int* M3) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M3Rows && col < M3Cols) {
        int index = row * M3Rows + col;
        M3[index] = M1[index] + M2[index];
    }
}

// CUDA kernel to add two matrices in parallel, utilizing both thread and block level parallelism, as well as shared memory
__global__ void matrixAdditionSharedMemory(int* M1, int* M2, int* M3) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ int sharedMemory1[16];
    __shared__ int sharedMemory2[16];

    int index = row * M3Rows + col;

    // Load data into shared memory
    sharedMemory1[threadIdx.x] = M1[index];
    sharedMemory2[threadIdx.x] = M2[index];

    __syncthreads();  // Ensure all threads have loaded data

    if (row < M3Rows && col < M3Cols) {
        M3[index] = sharedMemory1[threadIdx.x] + sharedMemory2[threadIdx.x];
    }
}

// Function to measure and record execution times to a file
void measureAndRecordExecutionTimes(
    const char* outputFileName,
    Timer timer,
    void (*kernel)(int*, int*, int*),
    int* M1, int* M2, int* M3,
    dim3 gridDim, dim3 blockDim
) {
    // Open a new file to write the result into
    FILE* outputFile = fopen(outputFileName, "w");
    if (outputFile == NULL) {
        perror("Unable to create the output file");
        return;
    }

    for (int i = 0; i < 100; i++) {
        // Measure execution time for MMV1Sequential
        beginTimer(timer);
        hipDeviceSynchronize();
        kernel <<<gridDim, blockDim>>>(M1, M2, M3);
        hipDeviceSynchronize();
        float time = endTimerReturnTime(timer);

        fprintf(outputFile, "%f ms\n", time);
    }

    // Close the output file
    fclose(outputFile);
}

int main() {
    // Timer measure time spent on a process
    Timer timer = createTimer();

    // Start the setup timer
    beginTimer(timer);

    // Define variables
    Matrix M1;
    Matrix M2;
    Matrix M3;

    // Create the matrix objects
    M1 = createMatrix(M1Rows, M1Cols);
    M2 = createMatrix(M2Rows, M2Cols);
    M3 = createMatrix(M3Rows, M3Cols);

    // Populate the matrices
    populateWithOnes(M1);
    populateWithOnes(M2);

    // Stop the setup timer
    endTimer(timer, "setup");

    // Start the data transfer timer (CPU -> GPU / Host -> Device)
    beginTimer(timer);

    // Allocate memory for matrices on the GPU
    int* device_M1, * device_M2, * device_M3;

    hipMalloc((void**)&device_M1, M1Rows * M1Cols * sizeof(int));
    hipMalloc((void**)&device_M2, M2Rows * M2Cols * sizeof(int));
    hipMalloc((void**)&device_M3, M3Rows * M3Cols * sizeof(int));

    // Copy input matrices from host to device
    hipMemcpy(device_M1, M1.data, M1Rows * M1Cols * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_M2, M2.data, M2Rows * M2Cols * sizeof(int), hipMemcpyHostToDevice);

    // Stop the data transfer timer (CPU -> GPU / Host -> Device)
    endTimer(timer, "data transfer (CPU -> GPU)");

    // Define block and grid dimensions for CUDA kernel
    dim3 blockDim(16, 16);

    if (M3Rows <= 16 && M3Cols <= 16) {
        blockDim = dim3(M3Cols, M3Rows);  // Use matrix size for smaller matrices
    }

    dim3 gridDim((M3Cols + blockDim.x - 1) / blockDim.x, (M3Rows + blockDim.y - 1) / blockDim.y);

    // Measure and record execution times
    measureAndRecordExecutionTimes("Test/MA1SequentialResults.txt", timer, matrixAdditionSequential,   device_M1, device_M2, device_M3, gridDim, blockDim);
    measureAndRecordExecutionTimes("Test/MA2ParallelV1.txt",        timer, matrixAdditionParallelV1,   device_M1, device_M2, device_M3, gridDim, blockDim);
    measureAndRecordExecutionTimes("Test/MA3ParallelV2.txt",        timer, matrixAdditionParallelV2,   device_M1, device_M2, device_M3, gridDim, blockDim);
    measureAndRecordExecutionTimes("Test/MASharedMemory.txt",       timer, matrixAdditionSharedMemory, device_M1, device_M2, device_M3, gridDim, blockDim);

    // Copy the result matrix from device to host
    hipMemcpy(M3.data, device_M3, M3Rows * M3Cols * sizeof(int), hipMemcpyDeviceToHost);

    // Deallocate memory on the GPU and CPU
    hipFree(device_M1);
    hipFree(device_M2);
    hipFree(device_M3);

    // Exit program
    return 0;
}