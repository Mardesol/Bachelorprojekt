#include "additionIntKernels.cu"

const bool printDebugMessages = false;

int main() {
    // Timer measure time spent on a process
    Timer timer = createTimer();

    // Start the setup timer
    beginTimer(timer);

    // Create the matrix objects
    MatrixI M1 = createMatrixInts(M1Rows, M1Cols);
    MatrixI M2 = createMatrixInts(M2Rows, M2Cols);
    MatrixI M3 = createMatrixInts(M3Rows, M3Cols);

    // Populate the matrices
    populateWithRandomInts(M1);
    populateWithRandomInts(M2);

    // Stop the setup timer
    endTimer(timer, "setup", printDebugMessages);

    // Start the data transfer timer (CPU -> GPU / Host -> Device)
    beginTimer(timer);

    // Declare the matrix objects to be stored on the device
    int* device_M1, * device_M2, * device_M3;

    // Allocate memory for matrices on the GPU
    hipMalloc((void**)&device_M1, M1Rows * M1Cols * sizeof(int));
    hipMalloc((void**)&device_M2, M2Rows * M2Cols * sizeof(int));
    hipMalloc((void**)&device_M3, M3Rows * M3Cols * sizeof(int));

    // Copy input matrices from host to device
    hipMemcpy(device_M1, M1.data, M1Rows * M1Cols * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_M2, M2.data, M2Rows * M2Cols * sizeof(int), hipMemcpyHostToDevice);

    // Stop the data transfer timer (CPU -> GPU / Host -> Device)
    endTimer(timer, "data transfer (CPU -> GPU)", printDebugMessages);

    // Define block and grid dimensions for CUDA kernel
    dim3 blockDim(32,32);
    dim3 gridDim((M3Cols + blockDim.x - 1) / blockDim.x, (M3Rows + blockDim.y - 1) / blockDim.y);

    // Time the matrix addition
    const char* kernelName;
    kernelName = "SharedMemory";                                                                                    // Should reflect the chosen kernel, to name output file accordingly
    beginTimer(timer);
    SharedMemory <<<gridDim, blockDim >>> (device_M1, device_M2, device_M3);                                        // Launch the CUDA kernel to perform matrix addition
    endTimer(timer, "matrix addition (GPU)", printDebugMessages);

    // Time transfer from device to host
    beginTimer(timer);                                                                                              // Start the data transfer timer (GPU -> CPU / Device -> Host)
    hipMemcpy(M3.data, device_M3, M3Rows * M3Cols * sizeof(int), hipMemcpyDeviceToHost);                          // Copy the result matrix from device to host
    endTimer(timer, "data transfer (GPU -> CPU)", printDebugMessages);                                              // Stop the data transfer timer (GPU -> CPU / Device -> Host)

    // Open a new file to write the result into
    char fileName[100];                                                                                             // Max length filename (Just needs to be long enough)
    sprintf(fileName, "Test/Addition_%s_Ints_Runtime_Matrix_Size_%dx%d.csv", kernelName, M3Rows, M3Cols);           // Customize filename to reflect size of result matrix
    FILE* outputFile = fopen(fileName, "w");
    if (outputFile == NULL) {
        perror("Unable to create the output file");
        return 1;
    }

    // Write host_M3 to the result file
    for (int i = 0; i < M3Rows; i++) {
        for (int j = 0; j < M3Cols; j++) {
            fprintf(outputFile, "%d ", M3.data[i * M3Rows + j]);
        }
        fprintf(outputFile, "\n");
    }

    // Close the result file
    fclose(outputFile);

    // Deallocate memory on the GPU and CPU
    hipFree(device_M1);
    hipFree(device_M2);
    hipFree(device_M3);

    // End program
    return 0;
}
