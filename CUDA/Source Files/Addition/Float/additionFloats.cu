#include "additionFloatsKernels.cu"

const bool printDebugMessages = false;

int main() {
    // Timer measures the time spent on a process
    Timer timer = createTimer();

    // Start the setup timer
    beginTimer(timer);

    // Create the matrix objects
    MatrixF M1 = createMatrixFloats(M1Rows, M1Cols);
    MatrixF M2 = createMatrixFloats(M2Rows, M2Cols);
    MatrixF M3 = createMatrixFloats(M3Rows, M3Cols);

    // Populate the matrices
    populateWithRandomFloats(M1);
    populateWithRandomFloats(M2);

    // Stop the setup timer
    endTimer(timer, "setup", printDebugMessages);

    // Start the data transfer timer (CPU -> GPU / Host -> Device)
    beginTimer(timer);

    // Declare the matrix objects to be stored on the device
    float* device_M1, * device_M2, * device_M3;

    // Allocate memory for matrices on the GPU
    hipMalloc((void**)&device_M1, M1Rows * M1Cols * sizeof(float));                                                
    hipMalloc((void**)&device_M2, M2Rows * M2Cols * sizeof(float));
    hipMalloc((void**)&device_M3, M3Rows * M3Cols * sizeof(float));

    // Copy input matrices from host to device
    hipMemcpy(device_M1, M1.data, M1Rows * M1Cols * sizeof(float), hipMemcpyHostToDevice);                        
    hipMemcpy(device_M2, M2.data, M2Rows * M2Cols * sizeof(float), hipMemcpyHostToDevice);

    // Stop the data transfer timer (CPU -> GPU / Host -> Device)
    endTimer(timer, "data transfer (CPU -> GPU)", printDebugMessages);

    // Define block and grid dimensions for CUDA kernel
    dim3 blockDim(32, 32);
    dim3 gridDim((M3Cols + blockDim.x - 1) / blockDim.x, (M3Rows + blockDim.y - 1) / blockDim.y);

    // Time the matrix addition
    const char* kernelName;
    kernelName = "SharedMemory";                                                                                    // Should reflect the chosen kernel, to name output file accordingly
    beginTimer(timer);                                                                              
    SharedMemory <<<gridDim, blockDim >>> (device_M1, device_M2, device_M3);                                        // Launch the CUDA kernel to perform matrix addition
    endTimer(timer, "matrix addition (GPU)", printDebugMessages);

    // Time transfer from device to host
    beginTimer(timer);                                                                                              // Start the data transfer timer (GPU -> CPU / Device -> Host)
    hipMemcpy(M3.data, device_M3, M3Rows * M3Cols * sizeof(float), hipMemcpyDeviceToHost);                        // Copy the result matrix from device to host
    endTimer(timer, "data transfer (GPU -> CPU)", printDebugMessages);                                              // Stop the data transfer timer (GPU -> CPU / Device -> Host)

    // Open a new file to write the result into
    char fileName[100];                                                                                             // Max length filename (Just needs to be long enough)
    sprintf(fileName, "Test/Addition_%s_Floats_Runtime_Matrix_Size_%dx%d.csv", kernelName, M3Rows, M3Cols);         // Customize filename to reflect size of result matrix
    FILE* outputFile = fopen(fileName, "w");
    if (outputFile == NULL) {
        perror("Unable to create the output file");
        return 1;
    }

    // Write host_M3 to the result file
    for (int i = 0; i < M3Rows; i++) {
        for (int j = 0; j < M3Cols; j++) {
            fprintf(outputFile, "%f ", M3.data[i * M3Rows + j]);
        }
        fprintf(outputFile, "\n");
    }

    // Close the result file
    fclose(outputFile);

    // Deallocate memory on the GPU and CPU
    hipFree(device_M1);
    hipFree(device_M2);
    hipFree(device_M3);

    // End the program
    return 0;
}