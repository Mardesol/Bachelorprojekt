#include "additionDoublesKernels.cu"
#include "..\..\Matrix\matrixCompatability.cu"

const bool printDebugMessages = false;

// Function to measure kernel execution time
float measureKernelExecutionTime(
    void (*kernel)(double*, double*, double*),
    double* M1, double* M2, double* M3,
    dim3 gridDim, dim3 blockDim
) {
    Timer timer = createTimer();
    beginTimer(timer);

    hipDeviceSynchronize();
    kernel <<<gridDim, blockDim >>> (M1, M2, M3);
    hipDeviceSynchronize();

    return endTimerReturnTime(timer);
}

// Function to measure execution times and store them in an array
void measureExecutionTimes(
    float* executionTimes,
    void (*kernel)(double*, double*, double*),
    double* M1, double* M2, double* M3,
    dim3 gridDim, dim3 blockDim
) {
    for (int i = 0; i < 100; i++) {
        // Measure execution time for the kernel
        float time = measureKernelExecutionTime(kernel, M1, M2, M3, gridDim, blockDim);
        executionTimes[i] = time;
    }
}

int main() {
    if (!additionCheck(M1Rows, M1Cols, M2Rows, M2Cols)) {
        perror("Matrices must have the same size");
        return 1;
    }
    // Timer measure time spent on a process
    Timer timer = createTimer();

    // Start the setup timer
    beginTimer(timer);

    // Create the matrix objects
    MatrixD M1 = createMatrixDoubles(M1Rows, M1Cols);
    MatrixD M2 = createMatrixDoubles(M2Rows, M2Cols);
    MatrixD M3 = createMatrixDoubles(M3Rows, M3Cols);

    // Populate the matrices
    populateWithRandomDoubles(M1);
    populateWithRandomDoubles(M2);

    // Stop the setup timer
    endTimer(timer, "setup", printDebugMessages);

    // Start the data transfer timer (CPU -> GPU / Host -> Device)
    beginTimer(timer);

    // Allocate memory for matrices on the GPU
    double* device_M1, * device_M2, * device_M3;

    hipMalloc((void**)&device_M1, M1Rows * M1Cols * sizeof(double));
    hipMalloc((void**)&device_M2, M2Rows * M2Cols * sizeof(double));
    hipMalloc((void**)&device_M3, M3Rows * M3Cols * sizeof(double));

    // Copy input matrices from host to device
    hipMemcpy(device_M1, M1.data, M1Rows * M1Cols * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_M2, M2.data, M2Rows * M2Cols * sizeof(double), hipMemcpyHostToDevice);

    // Stop the data transfer timer (CPU -> GPU / Host -> Device)
    endTimer(timer, "data transfer (CPU -> GPU)", printDebugMessages);

    // Define block and grid dimensions for CUDA kernel
    dim3 blockDim(16, 16);

    if (M3Rows <= 16 && M3Cols <= 16) {
        blockDim = dim3(M3Cols, M3Rows);  // Use matrix size for smaller matrices
    }

    dim3 gridDim((M3Cols + blockDim.x - 1) / blockDim.x, (M3Rows + blockDim.y - 1) / blockDim.y);

    // Create an array to store execution times for each kernel
    float executionTimes[4][100]; // 4 kernels, 100 executions each

    // Measure and record execution times for all kernels
    measureExecutionTimes(executionTimes[0], Sequential,    device_M1, device_M2, device_M3, gridDim, blockDim);
    measureExecutionTimes(executionTimes[1], Parallel,      device_M1, device_M2, device_M3, gridDim, blockDim);
    measureExecutionTimes(executionTimes[2], SharedMemory,  device_M1, device_M2, device_M3, gridDim, blockDim);

    // Copy the result matrix from device to host
    hipMemcpy(M3.data, device_M3, M3Rows * M3Cols * sizeof(double), hipMemcpyDeviceToHost);

    // Deallocate memory on the GPU and CPU
    hipFree(device_M1);
    hipFree(device_M2);
    hipFree(device_M3);

    // Open a new file to write the result into
    char fileName[100];                                                                                             // Max length filename (Just needs to be long enough)
    sprintf(fileName, "Test/Double_Execution_Times_Matrix_Size_%dx%d.csv", M3Rows, M3Cols);                            // Customize filename to reflect size of result matrix
    FILE* outputFile = fopen(fileName, "w");
    if (outputFile == NULL) {
        perror("Unable to create the output file");
        return 1;
    }

    // Write execution times to the output file in separate columns
    fprintf(outputFile, "Sequential,Parallel,SharedMemory\n");
    for (int i = 0; i < 100; i++) {
        fprintf(outputFile, "%lf,%lf,%lf\n",
            executionTimes[0][i],
            executionTimes[1][i],
            executionTimes[2][i]);
    }

    // Close the output file
    fclose(outputFile);

    // Exit program
    return 0;
}