#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include "..\Timer\timer.cu"
#include "..\Matrix\matrix.cu"

// CUDA kernel to add two matrices sequentially
__global__ void Sequential(float *M1, float *M2, float *M3, int rows, int cols)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            M3[rows * i + j] = M1[rows * i + j] + M2[rows * i + j];
        }
    }
}

// CUDA kernel to add two matrices in parallel
__global__ void Parallel(float *M1, float *M2, float *M3, int rows, int cols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols)
    {
        int index = row * rows + col;
        M3[index] = M1[index] + M2[index];
    }
}

// CUDA kernel to add two matrices in parallel using shared memory
__global__ void SharedMemory(float *M1, float *M2, float *M3, int rows, int cols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float sharedMemory1[32 * 32];
    __shared__ float sharedMemory2[32 * 32];

    int index = row * rows + col;
    int sharedIndex = threadIdx.y * blockDim.x + threadIdx.x;

    // Ensure the index is within the matrix dimensions before loading into shared memory
    if (index < rows * cols)
    {
        sharedMemory1[sharedIndex] = M1[index];
        sharedMemory2[sharedIndex] = M2[index];
    }

    __syncthreads(); // Ensure all threads have loaded data

    if (index < rows * cols)
    {
        M3[index] = sharedMemory1[sharedIndex] + sharedMemory2[sharedIndex];
    }
}