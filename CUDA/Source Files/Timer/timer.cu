#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"

#include "..\..\Header Files\timer.cuh"

struct Timer createTimer()
{
	struct Timer timer;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	timer.start = start;
	hipEventCreate(&stop);
	timer.stop = stop;
	return timer;
}

void beginTimer(Timer timer)
{	
	hipEventRecord(timer.start, 0);
}

void endTimer(Timer timer, const char* message)
{
	float timeElapsed;
	hipEventRecord(timer.stop, 0);
	hipEventSynchronize(timer.stop);
	hipEventElapsedTime(&timeElapsed, timer.start, timer.stop);
	printf("Time spent on %s: %f ms\n", message, timeElapsed);
}