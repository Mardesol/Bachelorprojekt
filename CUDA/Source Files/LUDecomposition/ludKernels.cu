#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include <math.h>

#include <stdio.h>
#include <stdlib.h>

#include "..\Timer\timer.cu"
#include "..\Matrix\matrix.cu"

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include <hipsolver.h>
#include <hip/hip_runtime_api.h>

__global__ void Sequential(float* A, int n) {
    // Loop over each row
    for (int i = 0; i < n; i++) {
        
        // Compute U elements (upper triangular part)
        for (int j = i; j < n; j++) {
            
            float sum = A[i * n + j];
            // Subtract the lower * upper products from sum
            for (int k = 0; k < i; k++) {
                sum -= A[i * n + k] * A[k * n + j];
            }
            A[i * n + j] = sum;
        }

        // Compute L elements (lower triangular part)
        for (int j = i + 1; j < n; j++) {
            
            float sum = A[j * n + i];
            // Subtract the lower * upper products from sum
            for (int k = 0; k < i; k++) {
                sum -= A[j * n + k] * A[k * n + i];
            }
            // Divide by the diagonal element
            A[j * n + i] = sum / A[i * n + i];
        }
    }
}

__global__ void Sequential_Partial_Pivoting(float* A, int n) {
    for (int i = 0; i < n; i++) {

        // Find pivot row
        int pivotRow = i;
        float maxVal = fabsf(A[i * n + i]);

        for (int p = i + 1; p < n; p++) {
            if (fabsf(A[p * n + i]) > maxVal) {
                maxVal = fabsf(A[p * n + i]);
                pivotRow = p;
            }
        }

        // Swap rows if needed
        if (pivotRow != i) {
            for (int j = 0; j < n; j++) {
                float temp = A[i * n + j];
                A[i * n + j] = A[pivotRow * n + j];
                A[pivotRow * n + j] = temp;
            }
        }

        __syncthreads();

        // Perform LUD
        for (int j = i; j < n; j++) {
            float sum = A[i * n + j];
            for (int k = 0; k < i; k++) {
                sum -= A[i * n + k] * A[k * n + j];
            }
            A[i * n + j] = sum;
        }

        for (int j = i + 1; j < n; j++) {
            float sum = A[j * n + i];
            for (int k = 0; k < i; k++) {
                sum -= A[j * n + k] * A[k * n + i];
            }
            A[j * n + i] = sum / A[i * n + i];
        }
    }
}

__global__ void New_Sequential(float* A, int n) {
    // Loop over each row - Must be done 1 at the time
    for (int i = 0; i < n; i++) {

        // Compute L elements (lower triangular part)
        for (int j = i + 1; j < n; j++) {
            A[j * n + i] = A[j * n + i] / A[i * n + i];
        }
        // Compute U elements (upper triangular part)
        for (int j = i + 1; j < n; j++) {
            for (int k = i + 1; k < n; k++) {
                A[j * n + k] = A[j * n + k] - A[i * n + k] * A[j * n + i];
            }
        }
    }
}

__global__ void New_Sequential_With_Partial_Pivoting(float* A, int n) {
    // Loop over each row - Must be done 1 at the time
    for (int i = 0; i < n; i++) {

        // Find pivot row
        int pivotRow = i;
        float maxVal = fabsf(A[i * n + i]);

        for (int p = i + 1; p < n; p++) {
            if (fabsf(A[p * n + i]) > maxVal) {
                maxVal = fabsf(A[p * n + i]);
                pivotRow = p;
            }
        }

        // Swap rows if needed
        if (pivotRow != i) {
            for (int j = 0; j < n; j++) {
                float temp = A[i * n + j];
                A[i * n + j] = A[pivotRow * n + j];
                A[pivotRow * n + j] = temp;
            }
        }

        __syncthreads();

        // Compute L elements (lower triangular part)
        for (int j = i + 1; j < n; j++) {
            A[j * n + i] = A[j * n + i] / A[i * n + i];
        }
        // Compute U elements (upper triangular part)
        for (int j = i + 1; j < n; j++) {
            for (int k = i + 1; k < n; k++) {
                A[j * n + k] = A[j * n + k] - A[i * n + k] * A[j * n + i];
            }
        }
    }
}

//Pivoting kernels
__global__ void PivotAndSwap(float* A, int* pivotIndices, int n, int i) {
    // Find the pivot: maximum element in the current column
    int maxIndex = i;
    float maxValue = abs(A[i * n + i]);
    for (int row = i + 1; row < n; ++row) {
        float value = abs(A[row * n + i]);
        if (value > maxValue) {
            maxIndex = row;
            maxValue = value;
        }
    }

    pivotIndices[i] = maxIndex;

    // Swap rows if necessary
    if (maxIndex != i) {
        for (int col = 0; col < n; ++col) {
            float temp = A[i * n + col];
            A[i * n + col] = A[maxIndex * n + col];
            A[maxIndex * n + col] = temp;
        }
    }
}

//Parallel kernels and main function

__global__ void ComputeLowerColumn(float* A, int n, int i) {
    int row = blockIdx.y * blockDim.y + threadIdx.y + i + 1;

    if (row < n) {
        A[row * n + i] = A[row * n + i] / A[i * n + i];
    }
}

__global__ void UpdateSubmatrix(float* A, int n, int i) {
    int row = blockIdx.y * blockDim.y + threadIdx.y + i + 1;
    int col = blockIdx.x * blockDim.x + threadIdx.x + i + 1;

    if (row < n && col < n) {
        A[row * n + col] = A[row * n + col] - A[i * n + col] * A[row * n + i];
    }
}

int* Parallel_Pivoted(float* A, int n, dim3 blockDim) {

    int* pivotIndices;
    hipMalloc(&pivotIndices, n * sizeof(int));

    // Loop over each row - Must be done 1 at the time
    for (int i = 0; i < n; i++) {

        // Invoke PivotAndSwap kernel
        PivotAndSwap << <1, 1 >> > (A, pivotIndices, n, i);
        hipDeviceSynchronize();

        //Dimensions of the submatrix below/to the right of element (i,i)
        int subMatrixDim = n - i - 1;

        // Calculates the L values for row j
        dim3 blockDimColumn(1, blockDim.y);
        dim3 gridDimColumn(1, (subMatrixDim + blockDim.x - 1) / blockDim.x);
        ComputeLowerColumn << <gridDimColumn, blockDimColumn >> > (A, n, i);
        // No difference in result by dropping these, but faster performance
        //hipDeviceSynchronize();

        dim3 gridDimSubmatrix((subMatrixDim + blockDim.x - 1) / blockDim.x, (subMatrixDim + blockDim.y - 1) / blockDim.y);
        UpdateSubmatrix << <gridDimSubmatrix, blockDim >> > (A, n, i);
        //hipDeviceSynchronize();

    }

    int* hostPivotIndices = (int*)malloc(n * sizeof(int));
    hipMemcpy(hostPivotIndices, pivotIndices, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(pivotIndices);
    return hostPivotIndices;
}

void Parallel(float* A, int n, dim3 blockDim) {

    // Loop over each row - Must be done 1 at the time
    for (int i = 0; i < n; i++) {

        //Dimensions of the submatrix below/to the right of element (i,i)
        int subMatrixDim = n - i - 1;

        // Calculates the L values for row j
        dim3 blockDimColumn(1, blockDim.y);
        dim3 gridDimColumn(1, (subMatrixDim + blockDim.x - 1) / blockDim.x);
        ComputeLowerColumn << <gridDimColumn, blockDimColumn >> > (A, n, i);
        hipDeviceSynchronize();

        dim3 gridDimSubmatrix((subMatrixDim + blockDim.x - 1) / blockDim.x, (subMatrixDim + blockDim.y - 1) / blockDim.y);
        UpdateSubmatrix << <gridDimSubmatrix, blockDim >> > (A, n, i);
        hipDeviceSynchronize();

    }
}

//Shared Memory kernels and main function

__global__ void PivotAndSwapShared(float* A, int* pivotIndices, int n, int i) {
    __shared__ float sharedRowA[32];
    __shared__ float sharedRowB[32];

    int tid = threadIdx.x;
    int maxIndex = pivotIndices[i]; // Assuming pivotIndices is precalculated

    // Ensure the thread ID is within the bounds of the row
    if (tid < n) {
        // Load the i-th and maxIndex-th rows into shared memory
        sharedRowA[tid] = A[i * n + tid];
        sharedRowB[tid] = A[maxIndex * n + tid];
    }

    // Wait for all threads in the block to load their elements
    __syncthreads();

    // Swap the elements in shared memory
    if (tid < n && maxIndex != i) {
        A[i * n + tid] = sharedRowB[tid];
        A[maxIndex * n + tid] = sharedRowA[tid];
    }
}

__global__ void ComputeLowerColumnShared(float* A, int n, int i) {
    __shared__ float pivotElement;
    pivotElement = A[i * n + i];

    int row = blockIdx.y * blockDim.y + threadIdx.y + i + 1;

    if (row < n) {
        A[row * n + i] /= pivotElement;
    }
}

__global__ void UpdateSubmatrixShared(float* A, int n, int i) {
    __shared__ float sharedRow[32];
    __shared__ float sharedCol[32];

    int row = blockIdx.y * blockDim.y + threadIdx.y + i + 1;
    int col = blockIdx.x * blockDim.x + threadIdx.x + i + 1;

    if (row < n && threadIdx.x == 0) {
        sharedCol[threadIdx.y] = A[row * n + i];
    }
    if (col < n && threadIdx.y == 0) {
        sharedRow[threadIdx.x] = A[i * n + col];
    }

    __syncthreads();

    if (row < n && col < n) {
        A[row * n + col] -= sharedRow[threadIdx.x] * sharedCol[threadIdx.y];
    }
}

int* SharedMemory_Pivoted(float* A, int n, dim3 blockDim) {
    int* pivotIndices;
    hipMalloc(&pivotIndices, n * sizeof(int));

    // Loop over each row - Must be done 1 at the time
    for (int i = 0; i < n; i++) {

        // Invoke PivotAndSwap kernel
        PivotAndSwap << <1, 1 >> > (A, pivotIndices, n, i);
        hipDeviceSynchronize();

        //Dimensions of the submatrix below/to the right of element (i,i)
        int subMatrixDim = n - i - 1;

        // Calculates the L values for row j
        dim3 blockDimColumn(1, blockDim.y);
        dim3 gridDimColumn(1, (subMatrixDim + blockDim.x - 1) / blockDim.x);
        ComputeLowerColumnShared << <gridDimColumn, blockDimColumn >> > (A, n, i);
        // No difference in result by dropping these, but faster performance
        //hipDeviceSynchronize();

        dim3 gridDimSubmatrix((subMatrixDim + blockDim.x - 1) / blockDim.x, (subMatrixDim + blockDim.y - 1) / blockDim.y);
        UpdateSubmatrixShared << <gridDimSubmatrix, blockDim >> > (A, n, i);
        //hipDeviceSynchronize();

    }

    int* hostPivotIndices = (int*)malloc(n * sizeof(int));
    hipMemcpy(hostPivotIndices, pivotIndices, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(pivotIndices);
    return hostPivotIndices;
}

