#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include <math.h>

#include <stdio.h>
#include <stdlib.h>

#include "..\Timer\timer.cu"
#include "..\Matrix\matrix.cu"

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include <hipsolver.h>
#include <hip/hip_runtime_api.h>

__global__ void Sequential(float* A, int n) {
    // Loop over each row
    for (int i = 0; i < n; i++) {
        
        // Compute U elements (upper triangular part)
        for (int j = i; j < n; j++) {
            
            float sum = A[i * n + j];
            // Subtract the lower * upper products from sum
            for (int k = 0; k < i; k++) {
                sum -= A[i * n + k] * A[k * n + j];
            }
            A[i * n + j] = sum;
        }

        // Compute L elements (lower triangular part)
        for (int j = i + 1; j < n; j++) {
            
            float sum = A[j * n + i];
            // Subtract the lower * upper products from sum
            for (int k = 0; k < i; k++) {
                sum -= A[j * n + k] * A[k * n + i];
            }
            // Divide by the diagonal element
            A[j * n + i] = sum / A[i * n + i];
        }
    }
}

__global__ void Sequential_Partial_Pivoting(float* A, int n) {
    for (int i = 0; i < n; i++) {

        // Find pivot row
        int pivotRow = i;
        float maxVal = fabsf(A[i * n + i]);

        for (int p = i + 1; p < n; p++) {
            if (fabsf(A[p * n + i]) > maxVal) {
                maxVal = fabsf(A[p * n + i]);
                pivotRow = p;
            }
        }

        // Swap rows if needed
        if (pivotRow != i) {
            for (int j = 0; j < n; j++) {
                float temp = A[i * n + j];
                A[i * n + j] = A[pivotRow * n + j];
                A[pivotRow * n + j] = temp;
            }
        }

        __syncthreads();

        // Perform LUD
        for (int j = i; j < n; j++) {
            float sum = A[i * n + j];
            for (int k = 0; k < i; k++) {
                sum -= A[i * n + k] * A[k * n + j];
            }
            A[i * n + j] = sum;
        }

        for (int j = i + 1; j < n; j++) {
            float sum = A[j * n + i];
            for (int k = 0; k < i; k++) {
                sum -= A[j * n + k] * A[k * n + i];
            }
            A[j * n + i] = sum / A[i * n + i];
        }
    }
}