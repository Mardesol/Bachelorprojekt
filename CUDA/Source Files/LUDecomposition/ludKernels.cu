#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include <math.h>

#include <stdio.h>
#include <stdlib.h>

#include "..\Timer\timer.cu"
#include "..\Matrix\matrix.cu"

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include <hipsolver.h>
#include <hip/hip_runtime_api.h>


__global__ void New_Sequential(float* A, int n) {
    // Loop over each row - Must be done 1 at the time
    for (int i = 0; i < n; i++) {

        // Compute L elements (lower triangular part)
        for (int j = i + 1; j < n; j++) {
            A[j * n + i] = A[j * n + i] / A[i * n + i];
        }
        // Compute U elements (upper triangular part)
        for (int j = i + 1; j < n; j++) {
            for (int k = i + 1; k < n; k++) {
                A[j * n + k] = A[j * n + k] - A[i * n + k] * A[j * n + i];
            }
        }
    }
}

__global__ void New_Sequential_With_Partial_Pivoting(float* A, int n) {
    // Loop over each row - Must be done 1 at the time
    for (int i = 0; i < n; i++) {

        // Find pivot row
        int pivotRow = i;
        float maxVal = fabsf(A[i * n + i]);

        for (int p = i + 1; p < n; p++) {
            if (fabsf(A[p * n + i]) > maxVal) {
                maxVal = fabsf(A[p * n + i]);
                pivotRow = p;
            }
        }

        // Swap rows if needed
        if (pivotRow != i) {
            for (int j = 0; j < n; j++) {
                float temp = A[i * n + j];
                A[i * n + j] = A[pivotRow * n + j];
                A[pivotRow * n + j] = temp;
            }
        }

        __syncthreads();

        // Compute L elements (lower triangular part)
        for (int j = i + 1; j < n; j++) {
            A[j * n + i] = A[j * n + i] / A[i * n + i];
        }
        // Compute U elements (upper triangular part)
        for (int j = i + 1; j < n; j++) {
            for (int k = i + 1; k < n; k++) {
                A[j * n + k] = A[j * n + k] - A[i * n + k] * A[j * n + i];
            }
        }
    }
}



//Pivoting kernels
__global__ void PivotAndSwap(float* A, int* pivotIndices, int n, int i) {
    // Find the pivot: maximum element in the current column
    int maxIndex = i;
    float maxValue = abs(A[i * n + i]);

    for (int row = i + 1; row < n; ++row) {
        float value = abs(A[row * n + i]);
        if (value > maxValue) {
            maxIndex = row;
            maxValue = value;
        }
    }

    pivotIndices[i] = maxIndex;

    // Swap rows if necessary
    if (maxIndex != i) {
        for (int col = 0; col < n; ++col) {
            float temp = A[i * n + col];
            A[i * n + col] = A[maxIndex * n + col];
            A[maxIndex * n + col] = temp;
        }
    }
}

__global__ void FindPivot(float* A, int* pivotIndices, int n, int i) {
    int maxIndex = i;
    float maxValue = abs(A[i * n + i]);

    for (int row = i + 1; row < n; ++row) {
        float value = abs(A[row * n + i]);
        if (value > maxValue) {
            maxIndex = row;
            maxValue = value;
        }
    }
    pivotIndices[i] = maxIndex;
}

__global__ void SwapRows(float* A, int* pivotIndices, int n, int i) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (int col = tid; col < n; col += blockDim.x * gridDim.x) {
        float temp = A[i * n + col];
        A[i * n + col] = A[pivotIndices[i] * n + col];
        A[pivotIndices[i] * n + col] = temp;
    }
}






//Parallel kernels and main function
__global__ void ComputeLowerColumn(float* A, int n, int i) {
    int row = blockIdx.y * blockDim.y + threadIdx.y + i + 1;

    if (row < n) {
        A[row * n + i] = A[row * n + i] / A[i * n + i];
    }
}

__global__ void UpdateSubmatrix(float* A, int n, int i) {
    int row = blockIdx.y * blockDim.y + threadIdx.y + i + 1;
    int col = blockIdx.x * blockDim.x + threadIdx.x + i + 1;

    if (row < n && col < n) {
        A[row * n + col] = A[row * n + col] - A[i * n + col] * A[row * n + i];
    }
}

int* Parallel_Pivoted(float* A, int n, dim3 blockDim) {

    int* pivotIndices;
    hipMalloc(&pivotIndices, n * sizeof(int));

    // Loop over each row - Must be done 1 at the time
    for (int i = 0; i < n; i++) {

        // Find pivot and swap

        //FindPivot << <1,1>> > (A, pivotIndices, n, i);
        dim3 blockDimRow(blockDim.x, 1);
        dim3 gridDimRow((n + blockDim.x - 1) / blockDim.x, 1);
        if (i == 0 || i == n - 1) { printf("launching pivot with a blockdim of %d, %d, and a gridDim of %d, %d\n", blockDim.x, 1, (n + blockDim.x - 1) / blockDim.x, 1); }
        //SwapRows << <gridDimRow, blockDimRow >> > (A, pivotIndices, n, i);
        PivotAndSwap << <gridDimRow, blockDimRow >> > (A, pivotIndices, n, i);
        //PivotAndSwap << <1, 1>> > (A, pivotIndices, n, i);
        hipDeviceSynchronize();

        //Dimensions of the submatrix below/to the right of element (i,i)
        int subMatrixDim = n - i - 1;

        // Calculates the L values for row j
        dim3 blockDimColumn(1, blockDim.y);
        dim3 gridDimColumn(1, (subMatrixDim + blockDim.x - 1) / blockDim.x);
        ComputeLowerColumn << <gridDimColumn, blockDimColumn >> > (A, n, i);
        // No difference in result by dropping these, but faster performance
        //hipDeviceSynchronize();

        dim3 gridDimSubmatrix((subMatrixDim + blockDim.x - 1) / blockDim.x, (subMatrixDim + blockDim.y - 1) / blockDim.y);
        UpdateSubmatrix << <gridDimSubmatrix, blockDim >> > (A, n, i);
        //hipDeviceSynchronize();

    }

    int* hostPivotIndices = (int*)malloc(n * sizeof(int));
    hipMemcpy(hostPivotIndices, pivotIndices, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(pivotIndices);
    return hostPivotIndices;
}







//Shared Memory kernels and main function

__global__ void PivotAndSwapShared(float* A, int* pivotIndices, int n, int i) {
    __shared__ float sharedRowA[32];
    __shared__ float sharedRowB[32];

    int tid = threadIdx.x;
    int maxIndex = pivotIndices[i]; // Assuming pivotIndices is precalculated

    // Ensure the thread ID is within the bounds of the row
    if (tid < n) {
        // Load the i-th and maxIndex-th rows into shared memory
        sharedRowA[tid] = A[i * n + tid];
        sharedRowB[tid] = A[maxIndex * n + tid];
    }

    // Wait for all threads in the block to load their elements
    __syncthreads();

    // Swap the elements in shared memory
    if (tid < n && maxIndex != i) {
        A[i * n + tid] = sharedRowB[tid];
        A[maxIndex * n + tid] = sharedRowA[tid];
    }
}

__global__ void ComputeLowerColumnShared(float* A, int n, int i) {
    __shared__ float pivotElement;
    pivotElement = A[i * n + i];

    int row = blockIdx.y * blockDim.y + threadIdx.y + i + 1;

    if (row < n) {
        A[row * n + i] /= pivotElement;
    }
}

__global__ void UpdateSubmatrixShared(float* A, int n, int i) {
    __shared__ float sharedRow[32];
    __shared__ float sharedCol[32];

    int row = blockIdx.y * blockDim.y + threadIdx.y + i + 1;
    int col = blockIdx.x * blockDim.x + threadIdx.x + i + 1;

    if (row < n && threadIdx.x == 0) {
        sharedCol[threadIdx.y] = A[row * n + i];
    }
    if (col < n && threadIdx.y == 0) {
        sharedRow[threadIdx.x] = A[i * n + col];
    }

    __syncthreads();

    if (row < n && col < n) {
        A[row * n + col] -= sharedRow[threadIdx.x] * sharedCol[threadIdx.y];
    }
}

int* SharedMemory_Pivoted(float* A, int n, dim3 blockDim) {
    int* pivotIndices;
    hipMalloc(&pivotIndices, n * sizeof(int));

    // Loop over each row - Must be done 1 at the time
    for (int i = 0; i < n; i++) {

        // Invoke PivotAndSwap kernel
        PivotAndSwap << <1, 1 >> > (A, pivotIndices, n, i);
        hipDeviceSynchronize();

        //Dimensions of the submatrix below/to the right of element (i,i)
        int subMatrixDim = n - i - 1;

        // Calculates the L values for row j
        dim3 blockDimColumn(1, blockDim.y);
        dim3 gridDimColumn(1, (subMatrixDim + blockDim.x - 1) / blockDim.x);
        ComputeLowerColumnShared << <gridDimColumn, blockDimColumn >> > (A, n, i);
        // No difference in result by dropping these, but faster performance
        //hipDeviceSynchronize();

        dim3 gridDimSubmatrix((subMatrixDim + blockDim.x - 1) / blockDim.x, (subMatrixDim + blockDim.y - 1) / blockDim.y);
        UpdateSubmatrixShared << <gridDimSubmatrix, blockDim >> > (A, n, i);
        //hipDeviceSynchronize();

    }

    int* hostPivotIndices = (int*)malloc(n * sizeof(int));
    hipMemcpy(hostPivotIndices, pivotIndices, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(pivotIndices);
    return hostPivotIndices;
}

