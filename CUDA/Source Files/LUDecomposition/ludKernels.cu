#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include <math.h>

#include <stdio.h>
#include <stdlib.h>

#include "..\Timer\timer.cu"
#include "..\Matrix\matrixFloats.cu"

__global__ void LUD_Sequential(float* A, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = i; j < n; j++) {
            float sum = A[i * n + j];
            for (int k = 0; k < i; k++) {
                sum -= A[i * n + k] * A[k * n + j];
            }
            A[i * n + j] = sum;
        }

        for (int j = i + 1; j < n; j++) {
            float sum = A[j * n + i];
            for (int k = 0; k < i; k++) {
                sum -= A[j * n + k] * A[k * n + i];
            }
            A[j * n + i] = sum / A[i * n + i];
        }
    }
}


__global__ void LUD_Sequential_Partial_Pivoting(float* A, int n) {
    for (int i = 0; i < n; i++) {

        // Find pivot row
        int pivotRow = i;
        float maxVal = fabsf(A[i * n + i]);

        for (int p = i + 1; p < n; p++) {
            if (fabsf(A[p * n + i]) > maxVal) {
                maxVal = fabsf(A[p * n + i]);
                pivotRow = p;
            }
        }

        // Swap rows if needed
        if (pivotRow != i) {
            for (int j = 0; j < n; j++) {
                float temp = A[i * n + j];
                A[i * n + j] = A[pivotRow * n + j];
                A[pivotRow * n + j] = temp;
            }
        }

        // Perform LUD
        for (int j = i; j < n; j++) {
            float sum = A[i * n + j];
            for (int k = 0; k < i; k++) {
                sum -= A[i * n + k] * A[k * n + j];
            }
            A[i * n + j] = sum;
        }

        for (int j = i + 1; j < n; j++) {
            float sum = A[j * n + i];
            for (int k = 0; k < i; k++) {
                sum -= A[j * n + k] * A[k * n + i];
            }
            A[j * n + i] = sum / A[i * n + i];
        }
    }
}


// __global__ void LUD_Parallel(float* A, int n) {

// }

__global__ void LUD_Block(float* A, int n) {
    
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x; 
    int ty = threadIdx.y;
    int bdx = blockDim.x;
    int bdy = blockDim.y;
    const int blockSize = 16;

    __shared__ float tile[blockSize][blockSize];

    // Load tile into shared memory
	int row = by * bdy + ty;
	int col = bx * bdx + tx;

    if(row < n && col < n) {
        tile[ty][tx] = A[row * n + col];
    }
    __syncthreads();

    // Perform LUD on the tile
    for (int k = 0; k < blockSize; k++) {
        if (tx == k && ty > k) {
            tile[ty][tx] /= tile[k][k];
        }
        __syncthreads();
        if (ty > k) {
            for (int j = k+1; j < blockSize; j++) {
                if (tx == j) {
                    tile[ty][tx] -= tile[ty][k] * tile[k][j];
                }
            }
        }
        __syncthreads();
    }

    // Write tile back to global memory
    if(row < n && col < n) {
        A[row * n + col] = tile[ty][tx];
    }
}
