#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include <math.h>

#include <stdio.h>
#include <stdlib.h>

#include "..\Timer\timer.cu"
#include "..\Matrix\matrix.cu"

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include <hipsolver.h>
#include <hip/hip_runtime_api.h>

void LUD_Sequential(float **A, int n) {
    // Loop over each row
    for (int i = 0; i < n; i++) {

        // Compute U elements (upper triangular part)
        for (int j = i; j < n; j++) {

            float sum = A[i][j];
            // Subtract the lower * upper products from sum
            for (int k = 0; k < i; k++) {
                sum -= A[i][k] * A[k][j];
            }
            A[i][j] = sum;
        }

        // Compute L elements (lower triangular part)
        for (int j = i + 1; j < n; j++) {
        
            float sum = A[j][i];
            // Subtract the lower * upper products from sum
            for (int k = 0; k < i; k++) {
                sum -= A[j][k] * A[k][i];
            }
            // Divide by the diagonal element
            A[j][i] = sum / A[i][i];
        }
    }
}

void LUD_Sequential(float **A, int n) {
    printf("before LUD \n");
    printf("n: %d \n", n);
    
    for (int i = 1; i <= n; i++) {
        printf("i: %d \n", i);
        
        for (int j = i; j <= n; j++) {
            printf("j1: %d \n", j);
            
            float sum = A[i][j];
            
            for (int k = 1; k < i; k++) {
                printf("k1: %d \n", k);
                sum -= A[i][k] * A[k][j];
            }
            A[i][j] = sum;
        }

        printf("after j loop \n");

        for (int j = i+1; j <= n; j++) {
            printf("j2: %d \n", j);
            float sum = A[j][i];
            
            for (int k = 1; k < i; k++) {
                printf("k2: %d \n", k);
                sum -= A[j][k] * A[k][i];
            }
            A[j][i] = sum / A[i][i];
        }
    }
    printf("after LUD \n");
}


void LUD_Sequential_Partial_Pivoting(float** A, int n) {
    for (int i = 1; i <= n; i++) {

        //Find pivot row                                    // Pivot row = row with the highest absolute value on the diagonal of the unworked rows
        int pivotRow = i;                                   //Set pivot row to current row
        float maxVal = fabs(A[i][i]);                       //Set max value to current rows diagonal
        for (int p = i + 1; p <= n; p++) {                  //Check if another row below has a higher absolut value on the diagonal
            if (fabs(A[p][i]) > maxVal) {
                maxVal = fabs(A[p][i]);                     //If yes, set that element to new max
                pivotRow = p;                               //And that row to the pivot row
            }
        }

        //Swap rows if needed
        if (pivotRow != i) {                                //Checks if current row is not the pivot row
            for (int j = 1; j <= n; j++) {                  //If not, swap the current row with the pivot row
                float temp = A[i][j];                       
                A[i][j] = A[pivotRow][j];
                A[pivotRow][j] = temp;
            }
        }

        //Perform LUD
        for (int j = i; j <= n; j++) {

            float sum = A[i][j];
            for (int k = 1; k < i; k++) {
                sum -= A[i][k] * A[k][j];
            }
            A[i][j] = sum;
        }

        for (int j = i + 1; j <= n; j++) {

            float sum = A[j][i];
            for (int k = 1; k < i; k++) {
                sum -= A[j][k] * A[k][i];
            }
            A[j][i] = sum / A[i][i];
        }
    }
}





//Copied from lud.cu
void LUD_cuSolver(float* device_A, int ADim, Timer timer) {
    hipsolverHandle_t handle;
    hipsolverDnCreate(&handle);

    int* d_pivot, * d_info;
    hipMalloc((void**)&d_pivot, ADim * sizeof(int));
    hipMalloc((void**)&d_info, sizeof(int));

    int lwork = 0;
    hipsolverDnSgetrf_bufferSize(handle, ADim, ADim, device_A, ADim, &lwork);
    float* work;
    hipMalloc((void**)&work, lwork * sizeof(float));

    beginTimer(timer);
    hipsolverDnSgetrf(handle, ADim, ADim, device_A, ADim, work, NULL, d_info);
    hipDeviceSynchronize();
    endTimer(timer, "cuSolver LUD (GPU)", printDebugMessages);

    // Clean up
    hipFree(d_pivot);
    hipFree(d_info);
    hipFree(work);
    hipsolverDnDestroy(handle);
}

void LUD_cuBLAS_Single(float* device_A, int ADim, Timer timer) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    int* d_info;
    hipMalloc((void**)&d_info, sizeof(int));

    // Wrap the pointer to device_A in an array for batched processing with batch size 1
    float* device_A_array[1] = { device_A };

    beginTimer(timer);
    hipblasSgetrfBatched(handle, ADim, device_A_array, ADim, NULL, d_info, 1);
    hipDeviceSynchronize();
    endTimer(timer, "cuBLAS LUD Single (GPU)", printDebugMessages);

    // Clean up
    hipFree(d_info);
    hipblasDestroy(handle);
}

