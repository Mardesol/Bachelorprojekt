#include "..\..\Header Files\matrixFloats.cuh"

#include <stdio.h>
#include <stdlib.h>

#include <hiprand/hiprand_kernel.h>

// Create a matrix on the host
MatrixF createMatrixFloats(int rows, int cols) {
    MatrixFloats matrix;
    matrix.rows = rows;
    matrix.cols = cols;

    // Allocate host memory for the matrix data
    matrix.data = (float*)malloc(rows * cols * sizeof(float));

    if (matrix.data == NULL) {
        printf("Memory allocation failed.\n");
        exit(1);
    }

    return matrix;
}

// Set all elements in the matrix to hold value 1.0f
void populateWithOnesFloats(MatrixFloats matrix) {
    for (int i = 0; i < matrix.rows; i++) {
        for (int j = 0; j < matrix.cols; j++) {
            matrix.data[i * matrix.cols + j] = 1.0f; // Change to float
        }
    }
}

// Generate random floats on the CPU using srand
void populateWithRandomFloats(MatrixFloats matrix) {
    srand(42);

    for (int i = 0; i < matrix.rows; i++) {
        for (int j = 0; j < matrix.cols; j++) {
            matrix.data[i * matrix.cols + j] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX); // Generate random float between 0 and 1
        }
    }
}
