#include "..\..\Header Files\matrixFloats.cuh"

#include <stdio.h>
#include <stdlib.h>

#include <hiprand/hiprand_kernel.h>

// Create a matrix on the host
MatrixF createMatrixF(int rows, int cols) {
    MatrixF matrix;
    matrix.rows = rows;
    matrix.cols = cols;

    // Allocate host memory for the matrix data
    matrix.data = (float*)malloc(rows * cols * sizeof(float));

    if (matrix.data == NULL) {
        printf("Memory allocation failed.\n");
        exit(1);
    }

    return matrix;
}

// Set all elements in the matrix to hold value 1.0f
void populateWithOnesF(MatrixF matrix) {
    for (int i = 0; i < matrix.rows; i++) {
        for (int j = 0; j < matrix.cols; j++) {
            matrix.data[i * matrix.cols + j] = 1.0f; // Change to float
        }
    }
}

// Generate random floats on the CPU using srand
void populateWithRandomFloats(MatrixF matrix) {
    srand(42);

    for (int i = 0; i < matrix.rows; i++) {
        for (int j = 0; j < matrix.cols; j++) {
            matrix.data[i * matrix.cols + j] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX); // Generate random float between 0 and 1
        }
    }
}

// Comparison for float data type
bool compareMatricesFloat(float* M1, float* M2, int MRows, int MCols) {
    for (int i = 0; i < MRows; i++) {
        for (int j = 0; j < MCols; j++) {
            if (M1[i * MCols + j] != M2[i * MCols + j]) {
                return false;  // Matrices do not match
            }
        }
    }
    return true;  // Matrices match
}
