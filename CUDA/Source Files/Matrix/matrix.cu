#include "hip/hip_runtime.h"
#include "..\..\Header Files\matrix.cuh"

#include <stdio.h>
#include <stdlib.h>

#include <hiprand/hiprand_kernel.h>

// Create a matrix on the host
Matrix createMatrix(int rows, int cols)
{
    Matrix matrix;
    matrix.rows = rows;
    matrix.cols = cols;

    // Allocate host memory for the matrix data
    matrix.data = (float *)malloc(rows * cols * sizeof(float));

    if (matrix.data == NULL)
    {
        printf("Memory allocation failed.\n");
        exit(1);
    }

    return matrix;
}

// Set all elements in the matrix to hold value 1.0f
void populateWithOnes(Matrix matrix)
{
    for (int i = 0; i < matrix.rows; i++)
    {
        for (int j = 0; j < matrix.cols; j++)
        {
            matrix.data[i * matrix.cols + j] = 1.0f; // Change to float
        }
    }
}

// Generate random floats on the CPU using srand
void populateWithRandomFloats(Matrix matrix)
{
    srand(42);

    for (int i = 0; i < matrix.rows; i++)
    {
        for (int j = 0; j < matrix.cols; j++)
        {
            //matrix.data[i * matrix.cols + j] = (float)rand() / RAND_MAX;
            matrix.data[i * matrix.cols + j] = (float)rand() / rand();
        }
    }
}

void printMatrixToFile(char *fileName, Matrix M)
{
    FILE *outputFile = fopen(fileName, "w");
    if (outputFile == NULL)
    {
        perror("Unable to create the output file");
        return;
    }

    // Print the matrix to the file
    for (int i = 0; i < M.rows; i++)
    {
        for (int j = 0; j < M.cols; j++)
        {
            fprintf(outputFile, "%f ", M.data[i * M.cols + j]);
        }
        fprintf(outputFile, "\n");
    }

    fclose(outputFile); // Close the file after writing
}

bool compareMatrices(Matrix M1, Matrix M2)
{
    const float ErrorMargin = (float)1;
    //const float ErrorMargin = 1e-6f;

    for (int i = 0; i < M1.rows; i++)
    {
        for (int j = 0; j < M1.cols; j++)
        {
            if (fabs(M1.data[i * M1.cols + j] - M2.data[i * M1.cols + j]) > ErrorMargin)
            {
                return false; // Matrices do not match
            }
        }
    }
    return true; // Matrices match
}

bool compareAndPrintDifferences(Matrix M1, Matrix M2, char* fileName) {
    const float ErrorMargin = (float)1;
    // const float ErrorMargin = 1e-6f;
    bool matricesMatch = true;

    // Create a matrix for the differences using the createMatrix function
    Matrix Differences = createMatrix(M1.rows, M1.cols);

    for (int i = 0; i < M1.rows; i++) {
        for (int j = 0; j < M1.cols; j++) {
            float diff = fabs(M1.data[i * M1.cols + j] - M2.data[i * M1.cols + j]);
            Differences.data[i * Differences.cols + j] = diff;

            if (diff > ErrorMargin) {
                matricesMatch = false;
            }
        }
    }

    printMatrixToFile(fileName, Differences);

    free(Differences.data);

    return matricesMatch;
}


Matrix twoDim_to_MatrixF(float** twoDim, int rows, int cols)
{
    Matrix matrix;
    matrix.rows = rows;
    matrix.cols = cols;

    // Allocate host memory for the matrix data
    matrix.data = (float *)malloc(rows * cols * sizeof(float));

    for(int i = 0; i < rows; ++i) {
        for(int j = 0; j < cols; ++j) {
            matrix.data[i * cols + j] = twoDim[i][j];
        }
    }

    return matrix;
}

float** MatrixF_to_twoDim(Matrix matrix) {
    float **twoDim = (float **)malloc(matrix.rows * sizeof(float *));
    for(int i = 0; i < matrix.rows; ++i) {
        twoDim[i] = (float *)malloc(matrix.cols * sizeof(float));
    }

    for(int i = 0; i < matrix.rows; ++i) {
        for(int j = 0; j < matrix.cols; ++j) {
            twoDim[i][j] = matrix.data[i * matrix.cols + j];
        }
    }
    return twoDim;
}

void initializeMatricesAndMemory(Matrix &M1, Matrix &M2, Matrix &M3, int M1Rows, int M1Cols, int M2Rows, int M2Cols, int M3Rows, int M3Cols)
{
    M1 = createMatrix(M1Rows, M1Cols);
    M2 = createMatrix(M2Rows, M2Cols);
    M3 = createMatrix(M2Rows, M2Cols);

    populateWithRandomFloats(M1);
    populateWithRandomFloats(M2);
}

void allocateMemoryOnGPU(float *&device_M1, float *&device_M2, float *&device_M3, size_t memorySize1, size_t memorySize2, size_t memorySize3)
{
    hipMalloc((void **)&device_M1, memorySize1);
    hipMalloc((void **)&device_M2, memorySize2);
    hipMalloc((void **)&device_M3, memorySize3);
}

void copyMatricesToGPU(const Matrix &M1, const Matrix &M2, float *device_M1, float *device_M2, size_t memorySize1, size_t memorySize2)
{
    hipMemcpy(device_M1, M1.data, memorySize1, hipMemcpyHostToDevice);
    hipMemcpy(device_M2, M2.data, memorySize2, hipMemcpyHostToDevice);
}

void freeMemory(float *device_M1, float *device_M2, float *device_M3, Matrix &M1, Matrix &M2, Matrix &M3)
{
    hipFree(device_M1);
    hipFree(device_M2);
    hipFree(device_M3);

    free(M1.data);
    free(M2.data);
    free(M3.data);
}