#include "..\..\Header Files\matrixInts.cuh"

#include <stdio.h>
#include <stdlib.h>

#include <hiprand/hiprand_kernel.h>

// Create a matrix on the host
Matrix createMatrix(int rows, int cols) {
    Matrix matrix;
    matrix.rows = rows;
    matrix.cols = cols;

    // Allocate host memory for the matrix data
    matrix.data = (int*)malloc(rows * cols * sizeof(int));

    if (matrix.data == NULL) {
        printf("Memory allocation failed.\n");
        exit(1);
    }

    return matrix;
}

// Set all elements in the matrix to hold value 1
void populateWithOnes(Matrix matrix) {
    for (int i = 0; i < matrix.rows; i++) {
        for (int j = 0; j < matrix.cols; j++) {
            matrix.data[i * matrix.cols + j] = 1;
        }
    }
}


// Generate random integers on the CPU using srand
void populateWithRandomInts(Matrix matrix) {
    srand(42);

    for (int i = 0; i < matrix.rows; i++) {
        for (int j = 0; j < matrix.cols; j++) {
            matrix.data[i * matrix.cols + j] = rand(); // rand() generates a random number based on the seed (CPU only)
        }
    }
}
