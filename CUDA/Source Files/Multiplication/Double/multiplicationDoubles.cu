#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include "..\..\Matrix\matrixDoubles.cu"  // Update to the header file for double matrices
#include "..\..\Timer\timer.cu"
#include "..\..\Matrix\matrixOperationsCPU.cu"
#include "..\..\Matrix\matrixCompatability.cu"

const int M1Rows = 200;
const int M1Cols = 200;
const int M2Rows = 200;
const int M2Cols = 200;

__global__ void MMV1Sequential(double* M1, double* M2, double* M3) {
    for (int i = 0; i < M1Rows; i++) {
        for (int j = 0; j < M2Cols; j++) {
            double sum = 0.0;
            for (int k = 0; k < M1Cols; k++) {
                sum += M1[i * M1Cols + k] * M2[k * M2Cols + j];
            }
            M3[i * M2Cols + j] = sum;
        }
    }
}

__global__ void MMV2Parallelism(double* M1, double* M2, double* M3) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M1Rows && col < M2Cols) {
        double sum = 0.0;

        for (int i = 0; i < M1Cols; i++) {
            sum += M1[row * M1Cols + i] * M2[i * M2Cols + col];
        }
        M3[row * M2Cols + col] = sum;
    }
}

__global__ void MMV3SharedMemoryAndTiling(double* M1, double* M2, double* M3) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Allocate shared memory
    __shared__ double sharedMemory1[16];
    __shared__ double sharedMemory2[16];

    double sum = 0.0;

    // Read into shared memory in a coalescing manner
    for (int i = 0; i < M1Cols; i += blockDim.x) {
        sharedMemory1[threadIdx.x] = (row < M1Rows && i + threadIdx.x < M1Cols) ? M1[row * M1Cols + i + threadIdx.x] : 0.0;
        sharedMemory2[threadIdx.y] = (i + threadIdx.y < M1Cols && col < M2Cols) ? M2[(i + threadIdx.y) * M2Cols + col] : 0.0;

        __syncthreads();

        // Perform the multiplication
        for (int j = 0; j < blockDim.x; j++) {
            sum += sharedMemory1[j] * sharedMemory2[j];
        }

        __syncthreads();
    }

    if (row < M1Rows && col < M2Cols) {
        M3[row * M2Cols + col] = sum;
    }
}

int main() {
    if (!multiplicationCheck(M1Cols, M2Rows)) {
        perror("Matrices must be compatible");
        return 1;
    }

    // Timer measure time spent on a process
    Timer timer = createTimer();

    // Start the setup timer
    beginTimer(timer);

    // Define variables
    MatrixD M1;  // Use MatrixDouble for double data type
    MatrixD M2;  // Use MatrixDouble for double data type
    MatrixD M3;  // Use MatrixDouble for double data type
    int M3Rows = M1Rows;
    int M3Cols = M2Cols;

    // Create the matrix objects
    M1 = createMatrixD(M1Rows, M1Cols);  // Use createMatrixDouble
    M2 = createMatrixD(M2Rows, M2Cols);  // Use createMatrixDouble
    M3 = createMatrixD(M3Rows, M3Cols);  // Use createMatrixDouble

    // Populate the matrices
    populateWithOnesD(M1);  // Use populateWithOnesDouble
    populateWithOnesD(M2);  // Use populateWithOnesDouble

    //Setup a CPU comparison matrix
    MatrixD MCPU = createMatrixD(M3Rows, M3Cols);
    additionDouble(M1.data, M2.data, MCPU.data, M3Rows, M3Cols);

    // Stop the setup timer
    endTimer(timer, "setup");

    // Start the data transfer timer (CPU -> GPU / Host -> Device)
    beginTimer(timer);

    // Create the matrix objects to be stored on the device
    double* device_M1, * device_M2, * device_M3;  // Change data type to double

    // Allocate memory for matrices on the GPU
    hipMalloc((void**)&device_M1, M1Rows * M1Cols * sizeof(double));  // Change data type to double
    hipMalloc((void**)&device_M2, M2Rows * M2Cols * sizeof(double));  // Change data type to double
    hipMalloc((void**)&device_M3, M3Rows * M3Cols * sizeof(double));  // Change data type to double

    // Copy data from host to device
    // The data is matrix 1 and 2
    hipMemcpy(device_M1, M1.data, M1Rows * M1Cols * sizeof(double), hipMemcpyHostToDevice);  // Change data type to double
    hipMemcpy(device_M2, M2.data, M2Rows * M2Cols * sizeof(double), hipMemcpyHostToDevice);  // Change data type to double

    // Stop the data transfer timer (CPU -> GPU / Host -> Device)
    endTimer(timer, "data transfer (CPU -> GPU)");

    // Define block and grid dimensions for CUDA kernel
    dim3 blockDim(16, 16);

    if (M3Rows <= 16 && M3Cols <= 16) {
        blockDim = dim3(M3Cols, M3Rows);  // Use matrix size for smaller matrices
    }

    dim3 gridDim((M3Cols + blockDim.x - 1) / blockDim.x, (M3Rows + blockDim.y - 1) / blockDim.y);

    // Start the matrix multiplication timer
    beginTimer(timer);

    // Launch the CUDA kernel to perform matrix multiplication
    hipDeviceSynchronize();
    MMV1Sequential << <gridDim, blockDim >> > (device_M1, device_M2, device_M3);
    hipDeviceSynchronize();

    // Stop the matrix multiplication timer
    endTimer(timer, "matrix multiplication (GPU)");

    // Start the data transfer timer (GPU -> CPU / Device -> Host)
    beginTimer(timer);

    // Copy the result matrix from device to host
    hipMemcpy(M3.data, device_M3, M3Rows * M3Cols * sizeof(double), hipMemcpyDeviceToHost);  // Change data type to double

    // Stop the data transfer timer (GPU -> CPU / Device -> Host)
    endTimer(timer, "data transfer (GPU -> CPU)");

    // Open a new file to write the result into
    FILE* outputFile = fopen("resultDoubles.txt", "w");
    if (outputFile == NULL) {
        perror("Unable to create the output file");
        return 1;
    }

    // Write host_M3 to the result file
    for (int i = 0; i < M3Rows; i++) {
        for (int j = 0; j < M3Cols; j++) {
            fprintf(outputFile, "%lf ", M3.data[i * M3Rows + j]);  // Change format specifier to %lf for double
        }
        fprintf(outputFile, "\n");
    }

    // Close the result file
    fclose(outputFile);

    //Validate result by comparing to CPU calculations
    bool valid = compareMatricesDouble(MCPU.data, M3.data, M3Rows, M3Cols);
    if (valid) {
        printf("Matrix multiplication results match!\n");
    }
    else {
        printf("Matrix multiplication results do not match.\n");
        // Write the matrices to text files for analysis
        FILE* outputFile1 = fopen("resultDoubleCPU.txt", "w");
        if (outputFile1 == NULL) {
            perror("Unable to create the output file");
            return 1;
        }

        // Write host_M3 to the result file
        for (int i = 0; i < M3Rows; i++) {
            for (int j = 0; j < M3Cols; j++) {
                fprintf(outputFile1, "%lf ", MCPU.data[i * M3Rows + j]);  // Change format specifier to %lf for double
            }
            fprintf(outputFile1, "\n");
        }

        // Close the result file
        fclose(outputFile1);

        FILE* outputFile2 = fopen("resultDoubleGPU.txt", "w");
        if (outputFile2 == NULL) {
            perror("Unable to create the output file");
            return 1;
        }

        // Write host_M3 to the result file
        for (int i = 0; i < M3Rows; i++) {
            for (int j = 0; j < M3Cols; j++) {
                fprintf(outputFile2, "%lf ", M3.data[i * M3Rows + j]);  // Change format specifier to %lf for double
            }
            fprintf(outputFile2, "\n");
        }

        // Close the result file
        fclose(outputFile2);
    }

    // Deallocate memory on the GPU and CPU
    hipFree(device_M1);
    hipFree(device_M2);
    hipFree(device_M3);

    // Exit program
    return 0;
}