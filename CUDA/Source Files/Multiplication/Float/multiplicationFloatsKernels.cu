#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include "..\..\Matrix\matrixFloats.cu"
#include "..\..\Timer\timer.cu"

__global__ void Sequential(float* M1, float* M2, float* M3) {
	for (int i = 0; i < M1Rows; i++) {
		for (int j = 0; j < M2Cols; j++) {
			float sum = 0.0f;
			for (int k = 0; k < M1Cols; k++) {
				sum += M1[i * M1Cols + k] * M2[k * M2Cols + j];
			}
			M3[i * M2Cols + j] = sum;
		}
	}
}

__global__ void Parallel(float* M1, float* M2, float* M3) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < M1Rows && col < M2Cols) {
		float sum = 0.0f;

		for (int i = 0; i < M1Cols; i++) {
			sum += M1[row * M1Cols + i] * M2[i * M2Cols + col];
		}
		M3[row * M2Cols + col] = sum;
	}
}

//__global__ void SharedMemoryAndTiling(float* M1, float* M2, float* M3) {
//	int row = blockIdx.y * blockDim.y + threadIdx.y;
//	int col = blockIdx.x * blockDim.x + threadIdx.x;
//
//	// Allocate shared memory
//	__shared__ float sharedMemory1[32 * 32];
//	__shared__ float sharedMemory2[32 * 32];
//
//	int sharedIndex1 = threadIdx.y * blockDim.x + threadIdx.x;
//	int sharedIndex2 = threadIdx.x * blockDim.y + threadIdx.y;
//
//	float sum = 0.0f;
//
//	// Read into shared memory in a coalescing manner
//	for (int i = 0; i < M1Cols; i += blockDim.x) {
//		sharedMemory1[sharedIndex1] = (row < M1Rows && i + threadIdx.x < M1Cols) ? M1[row * M1Cols + i + threadIdx.x] : 0.0f;
//		sharedMemory2[sharedIndex2] = (i + threadIdx.y < M1Cols && col < M2Cols) ? M2[(i + threadIdx.y) * M2Cols + col] : 0.0f;
//
//		__syncthreads();
//
//		// Perform the multiplication
//		for (int j = 0; j < blockDim.x; j++) {
//			sum += sharedMemory1[threadIdx.y * blockDim.x + j] * sharedMemory2[j * blockDim.y + threadIdx.x];
//		}
//
//		__syncthreads();
//	}
//
//	if (row < M1Rows && col < M2Cols) {
//		M3[row * M2Cols + col] = sum;
//	}
//}

__global__ void SharedMemoryAndTiling(float* M1, float* M2, float* M3) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ float sharedMemory1[256];
	__shared__ float sharedMemory2[256];

	int sharedIndex = threadIdx.y * blockDim.x + threadIdx.x;

	float sum = 0.0f;

	for (int i = 0; i < M1Cols; i += blockDim.x) {
		// Load M1 into shared memory
		if (row < M1Rows && (i + threadIdx.x) < M1Cols)
			sharedMemory1[sharedIndex] = M1[row * M1Cols + i + threadIdx.x];
		else
			sharedMemory1[sharedIndex] = 0;

		// Load M2 into shared memory
		if ((i + threadIdx.y) < M1Cols && col < M2Cols)
			sharedMemory2[sharedIndex] = M2[(i + threadIdx.y) * M2Cols + col];
		else
			sharedMemory2[sharedIndex] = 0;

		__syncthreads();

		// Tile multiplication
		int numIterations = (M1Cols - i > blockDim.x) ? blockDim.x : M1Cols - i;
		for (int j = 0; j < numIterations; j++) {
			sum += sharedMemory1[threadIdx.y * blockDim.x + j] * sharedMemory2[j * blockDim.x + threadIdx.x];
		}

		__syncthreads();
	}

	if (row < M1Rows && col < M2Cols) {
		M3[row * M2Cols + col] = sum;
	}
}