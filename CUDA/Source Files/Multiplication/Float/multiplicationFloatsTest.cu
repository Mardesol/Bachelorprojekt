#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include "..\..\Matrix\matrixFloats.cu"
#include "..\..\Timer\timer.cu"

const int M1Rows = 100;
const int M1Cols = 100;
const int M2Rows = 100;
const int M2Cols = 100;
const int M3Rows = M1Rows;
const int M3Cols = M2Cols;

__global__ void MMV1Sequential(float* M1, float* M2, float* M3) {

	for (int i = 0; i < M1Rows; i++) {
		for (int j = 0; j < M2Cols; j++) {
			float sum = 0.0f;
			for (int k = 0; k < M1Cols; k++) {
				sum += M1[i * M1Cols + k] *
					M2[k * M2Cols + j];
			}
			M3[i * M2Cols + j] = sum;
		}
	}
}

__global__ void MMV2Parallelism(float* M1, float* M2, float* M3) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < M1Rows && col < M2Cols) {
		float sum = 0.0f;

		for (int i = 0; i < M1Cols; i++) {
			sum += M1[row * M1Cols + i] *
				M2[i * M2Cols + col];
		}
		M3[row * M2Cols + col] = sum;
	}
}

__global__ void MMV3SharedMemoryAndTiling(float* M1, float* M2, float* M3) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// Allocate shared memory
	__shared__ float sharedMemory1[16];
	__shared__ float sharedMemory2[16];

	float sum = 0.0f;

	// Read into shared memory in a coalescing manner
	for (int i = 0; i < M1Cols; i += blockDim.x) {
		sharedMemory1[threadIdx.x] = (row < M1Rows && i + threadIdx.x < M1Cols) ? M1[row * M1Cols + i + threadIdx.x] : 0.0f;
		sharedMemory2[threadIdx.y] = (i + threadIdx.y < M1Cols && col < M2Cols) ? M2[(i + threadIdx.y) * M2Cols + col] : 0.0f;

		__syncthreads();

		// Perform the multiplication
		for (int j = 0; j < blockDim.x; j++) {
			sum += sharedMemory1[j] * sharedMemory2[j];
		}

		__syncthreads();
	}

	if (row < M1Rows && col < M2Cols) {
		M3[row * M2Cols + col] = sum;
	}
}

// Function to measure kernel execution time
float measureKernelExecutionTime(
	void (*kernel)(float*, float*, float*),
	float* M1, float* M2, float* M3,
	dim3 gridDim, dim3 blockDim
) {
	Timer timer = createTimer();
	beginTimer(timer);

	hipDeviceSynchronize();
	kernel << <gridDim, blockDim >> > (M1, M2, M3);
	hipDeviceSynchronize();

	return endTimerReturnTime(timer);
}

// Function to measure execution times and store them in an array
void measureExecutionTimes(
	float* executionTimes,
	void (*kernel)(float*, float*, float*),
	float* M1, float* M2, float* M3,
	dim3 gridDim, dim3 blockDim
) {
	for (int i = 0; i < 100; i++) {
		// Measure execution time for the kernel
		float time = measureKernelExecutionTime(kernel, M1, M2, M3, gridDim, blockDim);
		executionTimes[i] = time;
	}
}

int main() {
	// Timer measure time spent on a process
	Timer timer = createTimer();

	// Start the setup timer
	beginTimer(timer);

	// Define variables
	MatrixF M1;
	MatrixF M2;
	MatrixF M3;

	// Create the matrix objects
	M1 = createMatrixF(M1Rows, M1Cols);
	M2 = createMatrixF(M2Rows, M2Cols);
	M3 = createMatrixF(M3Rows, M3Cols);

	// Populate the matrices
	populateWithOnesFloats(M1);
	populateWithOnesFloats(M2);

	// Stop the setup timer
	endTimer(timer, "setup");

	// Start the data transfer timer (CPU -> GPU / Host -> Device)
	beginTimer(timer);

	// Create the matrix objects to be stored on the device
	float* device_M1, * device_M2, * device_M3;

	// Allocate memory for matrices on the GPU
	hipMalloc((void**)&device_M1, M1Rows * M1Cols * sizeof(float));
	hipMalloc((void**)&device_M2, M2Rows * M2Cols * sizeof(float));
	hipMalloc((void**)&device_M3, M3Rows * M3Cols * sizeof(float));

	// Copy data from host to device
	hipMemcpy(device_M1, M1.data, M1Rows * M1Cols * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_M2, M2.data, M2Rows * M2Cols * sizeof(float), hipMemcpyHostToDevice);

	// Stop the data transfer timer (CPU -> GPU / Host -> Device)
	endTimer(timer, "data transfer (CPU -> GPU)");

	// Define block and grid dimensions for CUDA kernel
	dim3 blockDim(16, 16);

	if (M3Rows <= 16 && M3Cols <= 16) {
		blockDim = dim3(M3Cols, M3Rows);  // Use matrix size for smaller matrices
	}

	dim3 gridDim((M3Cols + blockDim.x - 1) / blockDim.x, (M3Rows + blockDim.y - 1) / blockDim.y);


	// Create an array to store execution times for each kernel
	float executionTimes[3][100]; // 3 kernels, 100 executions each

	// Measure and record execution times
	measureExecutionTimes(executionTimes[0], MMV1Sequential,				device_M1, device_M2, device_M3, gridDim, blockDim);
	measureExecutionTimes(executionTimes[1], MMV2Parallelism,				device_M1, device_M2, device_M3, gridDim, blockDim);
	measureExecutionTimes(executionTimes[2], MMV3SharedMemoryAndTiling,		device_M1, device_M2, device_M3, gridDim, blockDim);

	// Copy the result matrix from device to host
	hipMemcpy(M3.data, device_M3, M3Rows * M3Cols * sizeof(int), hipMemcpyDeviceToHost);

	// Deallocate memory on the GPU and CPU
	hipFree(device_M1);
	hipFree(device_M2);
	hipFree(device_M3);

	// Open the output file for writing in append mode
	FILE* outputFile = fopen("Test/Floats-ExecutionTimes.csv", "a");
	if (outputFile == NULL) {
		perror("Unable to open the output file");
		return 1;
	}

	// Write execution times to the output file in separate columns
	fprintf(outputFile, "Sequential,Parallel,SharedMemoryAndTilling\n");
	for (int i = 0; i < 100; i++) {
		fprintf(outputFile, "%f,%f,%f\n",
			executionTimes[0][i],
			executionTimes[1][i],
			executionTimes[2][i]);
	}

	// Close the output file
	fclose(outputFile);

	// Exit program
	return 0;
}

