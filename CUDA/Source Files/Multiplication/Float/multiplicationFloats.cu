#include "multiplicationFloatsKernels.cu"
#include "..\..\Matrix\matrixOperationsCPU.cu"
#include "..\..\Matrix\matrixCompatability.cu"

const bool printDebugMessages = false;

int main(int argc, char *argv[])
{
    if (!isCompatibleForMultiplication(M1Cols, M2Rows))
    {
        perror("Matrices must be compatible");
        return 1;
    }

    // Timer measure time spent on a process
    Timer timer = createTimer();

    beginTimer(timer);
    MatrixF M1, M2, M3;
    float *device_M1, *device_M2, *device_M3;
    initializeMatricesAndMemory(M1, M2, M3);
    allocateMemoryOnGPU(device_M1, device_M2, device_M3);
    copyMatricesToGPU(M1, M2, device_M1, device_M2);
    endTimer(timer, "initialize matrices on CPU and GPU", printDebugMessages);

    // Define block and grid dimensions for CUDA kernel
    dim3 blockDim(16, 16);

    if (M3Rows <= 16 && M3Cols <= 16)
    {
        blockDim = dim3(M3Cols, M3Rows); // Use matrix size for smaller matrices
    }

    dim3 gridDim((M3Cols + blockDim.x - 1) / blockDim.x, (M3Rows + blockDim.y - 1) / blockDim.y);

    // Convert the command line argument to an integer
    int choice = atoi(argv[1]);

    const char *kernelName;

    if (choice == 1)
    {
        // Time the matrix multiplication
        kernelName = "Sequential"; // Should reflect the chosen kernel, to name output file accordingly
        beginTimer(timer);
        Sequential<<<gridDim, blockDim>>>(device_M1, device_M2, device_M3); // Launch the CUDA kernel to perform matrix addition
        endTimer(timer, "matrix multiplication (GPU)", printDebugMessages);
    }
    else if (choice == 2)
    {
        // Time the matrix multiplication
        kernelName = "Parallel"; // Should reflect the chosen kernel, to name output file accordingly
        beginTimer(timer);
        Parallel<<<gridDim, blockDim>>>(device_M1, device_M2, device_M3); // Launch the CUDA kernel to perform matrix addition
        endTimer(timer, "matrix multiplication (GPU)", printDebugMessages);
    }
    else if (choice == 3)
    {
        // Time the matrix multiplication
        kernelName = "SharedMemoryAndTiling"; // Should reflect the chosen kernel, to name output file accordingly
        beginTimer(timer);
        hipDeviceSynchronize();
        SharedMemoryAndTiling<<<gridDim, blockDim>>>(device_M1, device_M2, device_M3); // Launch the CUDA kernel to perform matrix addition
        hipDeviceSynchronize();
        endTimer(timer, "matrix multiplication (GPU)", printDebugMessages);
    }

    // Copy the result matrix from device to host
    hipMemcpy(M3.data, device_M3, memorySize3, hipMemcpyDeviceToHost);

    // Open a new file to write the result into
    char fileName[100];                                                                                           // Max length filename (Just needs to be long enough)
    sprintf(fileName, "Test/Multiplication_%s_Floats_Runtime_Matrix_Size_%dx%d.csv", kernelName, M3Rows, M3Cols); // Customize filename to reflect size of result matrix
    printMatrixToFileFloats(fileName, M3);

    // Setup a CPU comparison matrix
    MatrixF MCPU = createMatrixFloats(M3Rows, M3Cols);
    multiplicationFloats(M1, M2, MCPU);

    // Validate result by comparing to CPU calculations
    bool valid = compareMatricesFloats(MCPU, M3);
    if (valid)
    {
        printf("Matrix multiplication results match!\n");
    }
    else
    {
        printf("Matrix multiplication results do not match.\n");
        // Write the CPU matrix to text file for analysis
        char fileNameCPU[100];
        sprintf(fileNameCPU, "resultsIntsCPU.txt");

        printMatrixToFileFloats(fileNameCPU, MCPU);
    }

    freeMemory(device_M1, device_M2, device_M3, M1, M2, M3);

    // Exit program
    return 0;
}