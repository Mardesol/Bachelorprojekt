#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include "..\..\Matrix\matrixInts.cu"
#include "..\..\Timer\timer.cu"

__global__ void Sequential(int* M1, int* M2, int* M3) {
	for (int i = 0; i < M1Rows; i++) {
		for (int j = 0; j < M2Cols; j++) {
			int sum = 0;
			for (int k = 0; k < M1Cols; k++) {
				sum += M1[i * M1Cols + k] * M2[k * M2Cols + j];
			}
			M3[i * M2Cols + j] = sum;
		}
	}
}

__global__ void Parallel(int* M1, int* M2, int* M3) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < M1Rows && col < M2Cols) {
		int sum = 0;

		for (int i = 0; i < M1Cols; i++) {
			sum += M1[row * M1Cols + i] * M2[i * M2Cols + col];
		}
		M3[row * M2Cols + col] = sum;
	}
}

//__global__ void SharedMemoryAndTiling(int* M1, int* M2, int* M3) {
//	int row = blockIdx.y * blockDim.y + threadIdx.y;
//	int col = blockIdx.x * blockDim.x + threadIdx.x;
//
//	// Allocate shared memory
//	__shared__ int sharedMemory1[256];
//	__shared__ int sharedMemory2[256];
//
//	int sharedIndex1 = threadIdx.y * blockDim.x + threadIdx.x;
//	int sharedIndex2 = threadIdx.x * blockDim.y + threadIdx.y;
//
//	int sum = 0;
//
//	// Read into shared memory in a coalescing manner
//	for (int i = 0; i < M1Cols; i += blockDim.x) {
//		sharedMemory1[sharedIndex1] = (row < M1Rows && i + threadIdx.x < M1Cols) ? M1[row * M1Cols + i + threadIdx.x] : 0;
//		sharedMemory2[sharedIndex2] = (i + threadIdx.y < M1Cols && col < M2Cols) ? M2[(i + threadIdx.y) * M2Cols + col] : 0;
//
//		__syncthreads();
//
//		// Perform the multiplication
//		for (int j = 0; j < blockDim.x; j++) {	
//			sum += sharedMemory1[threadIdx.y * blockDim.x + j] * sharedMemory2[j * blockDim.y + threadIdx.x];
//		}
//
//		__syncthreads();
//	}
//
//	if (row < M1Rows && col < M2Cols) {
//		M3[row * M2Cols + col] = sum;
//	}
//}

__global__ void SharedMemoryAndTiling(int* M1, int* M2, int* M3) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ int sharedMemory1[256];
    __shared__ int sharedMemory2[256];

    int sharedIndex = threadIdx.y * blockDim.x + threadIdx.x;

    int sum = 0;

    for (int i = 0; i < M1Cols; i += blockDim.x) {
        // Load M1 into shared memory
        if (row < M1Rows && (i + threadIdx.x) < M1Cols)
            sharedMemory1[sharedIndex] = M1[row * M1Cols + i + threadIdx.x];
        else
            sharedMemory1[sharedIndex] = 0;

        // Load M2 into shared memory
        if ((i + threadIdx.y) < M1Cols && col < M2Cols)
            sharedMemory2[sharedIndex] = M2[(i + threadIdx.y) * M2Cols + col];
        else
            sharedMemory2[sharedIndex] = 0;

        __syncthreads();

        // Tile multiplication
        int numIterations = (M1Cols - i > blockDim.x) ? blockDim.x : M1Cols - i;
        for (int j = 0; j < numIterations; j++) {
            sum += sharedMemory1[threadIdx.y * blockDim.x + j] * sharedMemory2[j * blockDim.x + threadIdx.x];
        }

        __syncthreads();
    }

    if (row < M1Rows && col < M2Cols) {
        M3[row * M2Cols + col] = sum;
    }
}

