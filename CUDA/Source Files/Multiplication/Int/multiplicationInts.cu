#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include "..\..\Matrix\matrixInts.cu"
#include "..\..\Timer\timer.cu"
#include "..\..\Matrix\matrixOperationsCPU.cu"
#include "..\..\Matrix\matrixCompatability.cu"

const int M1Rows = 200;
const int M1Cols = 200;
const int M2Rows = 200;
const int M2Cols = 200;

__global__ void MMV1Sequential(int* M1, int* M2, int* M3) {
	
	for (int i = 0; i < M1Rows; i++) {
		for (int j = 0; j < M2Cols; j++) {
			int sum = 0;
			for (int k = 0; k < M1Cols; k++) {
				sum += M1[i * M1Cols + k] * 
					   M2[k * M2Cols + j];
			}
			M3[i * M2Cols + j] = sum;
		}
	}
}

__global__ void MMV2Parallelism(int* M1, int* M2, int* M3) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < M1Rows && col < M2Cols) {
		int sum = 0;

		for (int i = 0; i < M1Cols; i++) {
			sum += M1[row * M1Cols + i] * 
				   M2[i * M2Cols + col];
		}
		M3[row * M2Cols + col] = sum;
	}
}

__global__ void MMV3SharedMemoryAndTiling(int* M1, int* M2, int* M3) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// Allocate shared memory
	__shared__ int sharedMemory1[16];
	__shared__ int sharedMemory2[16];

	int sum = 0;

	// Read into shared memory in a coalescing manner
	for (int i = 0; i < M1Cols; i += blockDim.x) {
		sharedMemory1[threadIdx.x] = (row < M1Rows && i + threadIdx.x < M1Cols) ? M1[row * M1Cols + i + threadIdx.x] : 0;
		sharedMemory2[threadIdx.y] = (i + threadIdx.y < M1Cols && col < M2Cols) ? M2[(i + threadIdx.y) * M2Cols + col] : 0;
		
		__syncthreads();

		// Perform the multiplication
		for (int j = 0; j < blockDim.x; j++) {
			sum += sharedMemory1[j] * sharedMemory2[j];
		}

		__syncthreads();
	}

	if (row < M1Rows && col < M2Cols) {
		M3[row * M2Cols + col] = sum;
	}
}

int main() {
	if (!multiplicationCheck(M1Cols, M2Rows)) {
		perror("Matrices must be compatible");
		return 1;
	}

	// Timer measure time spent on a process
	Timer timer = createTimer();

	// Start the setup timer
	beginTimer(timer);

	// Define variables
	MatrixI M1;
	MatrixI M2;
	MatrixI M3;
	int M3Rows = M1Rows;
	int M3Cols = M2Cols;

	// Create the matrix objects
	M1 = createMatrixI(M1Rows, M1Cols);
	M2 = createMatrixI(M2Rows, M2Cols);
	M3 = createMatrixI(M3Rows, M3Cols);

	// Populate the matrices
	populateWithOnesI(M1);
	populateWithOnesI(M2);

	//Setup a CPU comparison matrix
	MatrixI MCPU = createMatrixI(M3Rows, M3Cols);
	additionInt(M1.data, M2.data, MCPU.data, M3Rows, M3Cols);

	// Stop the setup timer
	endTimer(timer, "setup");

	// Start the data transfer timer (CPU -> GPU / Host -> Device)
	beginTimer(timer);

	// Create the matrix objects to be stored on the device
	int* device_M1, * device_M2, * device_M3;

	// Allocate memory for matrices on the GPU
	hipMalloc((void**)&device_M1, M1Rows * M1Cols * sizeof(int));
	hipMalloc((void**)&device_M2, M2Rows * M2Cols * sizeof(int));
	hipMalloc((void**)&device_M3, M3Rows * M3Cols * sizeof(int));

	// Copy data from host to device
	// The data is matrix 1 and 2
	hipMemcpy(device_M1, M1.data, M1Rows * M1Cols * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_M2, M2.data, M2Rows * M2Cols * sizeof(int), hipMemcpyHostToDevice);

	// Stop the data transfer timer (CPU -> GPU / Host -> Device)
	endTimer(timer, "data transfer (CPU -> GPU)");

	// Define block and grid dimensions for CUDA kernel
	dim3 blockDim(16,16);

	if (M3Rows <= 16 && M3Cols <= 16) {
		blockDim = dim3(M3Cols, M3Rows);  // Use matrix size for smaller matrices
	}

	dim3 gridDim((M3Cols + blockDim.x - 1) / blockDim.x, (M3Rows + blockDim.y - 1) / blockDim.y);

	// Start the matrix addition timer
	beginTimer(timer);

	// Launch the CUDA kernel to perform matrix multiplication
	hipDeviceSynchronize();
	MMV1Sequential <<<gridDim, blockDim>>>(device_M1, device_M2, device_M3);
	hipDeviceSynchronize();

	// Stop the matrix multiplication timer
	endTimer(timer, "matrix multiplication (GPU)");

	// Start the data transfer timer (GPU -> CPU / Device -> Host)
	beginTimer(timer);

	// Copy the result matrix from device to host
	hipMemcpy(M3.data, device_M3, M3Rows * M3Cols * sizeof(int), hipMemcpyDeviceToHost);

	// Stop the data transfer timer (GPU -> CPU / Device -> Host)
	endTimer(timer, "data transfer (GPU -> CPU)");

	// Open a new file to write the result into
	FILE* outputFile = fopen("result.txt", "w");
	if (outputFile == NULL) {
		perror("Unable to create the output file");
		return 1;
	}

	// Write host_M3 to the result file
	for (int i = 0; i < M3Rows; i++) {
		for (int j = 0; j < M3Cols; j++) {
			fprintf(outputFile, "%d ", M3.data[i * M3Rows + j]);
		}
		fprintf(outputFile, "\n");
	}

	// Close the result file
	fclose(outputFile);

	//Validate result by comparing to CPU calculations
	bool valid = compareMatricesInt(MCPU.data, M3.data, M3Rows, M3Cols);
	if (valid) {
		printf("Matrix multiplication results match!\n");
	}
	else {
		printf("Matrix multiplication results do not match.\n");
		// Write the matrices to text files for analysis
		FILE* outputFile1 = fopen("resultIntsCPU.txt", "w");
		if (outputFile1 == NULL) {
			perror("Unable to create the output file");
			return 1;
		}

		// Write host_M3 to the result file
		for (int i = 0; i < M3Rows; i++) {
			for (int j = 0; j < M3Cols; j++) {
				fprintf(outputFile1, "%d ", MCPU.data[i * M3Rows + j]);  // Change format specifier to %lf for double
			}
			fprintf(outputFile1, "\n");
		}

		// Close the result file
		fclose(outputFile1);

		FILE* outputFile2 = fopen("resultIntsGPU.txt", "w");
		if (outputFile2 == NULL) {
			perror("Unable to create the output file");
			return 1;
		}

		// Write host_M3 to the result file
		for (int i = 0; i < M3Rows; i++) {
			for (int j = 0; j < M3Cols; j++) {
				fprintf(outputFile2, "%d ", M3.data[i * M3Rows + j]);  // Change format specifier to %lf for double
			}
			fprintf(outputFile2, "\n");
		}

		// Close the result file
		fclose(outputFile2);
	}

	// Deallocate memory on the GPU and CPU
	hipFree(device_M1);
	hipFree(device_M2);
	hipFree(device_M3);

	// Exit program
	return 0;
}

