#include "ludKernels.cu"
#include "..\Matrix\matrixOperationsCPU.cu"

// extern "C" {
//     void LUD_Sequential(float **A, int n);
//     void LUD_Sequential_Partial_Pivoting(float** A, int n);
// }



const bool printDebugMessages = true;
const size_t FILENAME_MAX_LENGTH = 256;

const char *executeChosenKernel(int KernelNumToPerform, float *device_A, int ADim, Timer timer)
{
    dim3 blockDim(32, 32);
    dim3 gridDim((ADim + blockDim.x - 1) / blockDim.x, (ADim + blockDim.y - 1) / blockDim.y);
    const char *kernelName;

    switch (KernelNumToPerform)
    {
    case 1:
        kernelName = "Sequential LUD (GPU)";
        beginTimer(timer);
        LUD_Sequential<<<gridDim, blockDim>>>(device_A, ADim);
        endTimer(timer, "Sequential LUD (GPU)", printDebugMessages);
        break;
    case 2:
        kernelName = "Sequential LUD with pivoting (GPU)";
        beginTimer(timer);
        LUD_Sequential_Partial_Pivoting<<<gridDim, blockDim>>>(device_A, ADim);
        endTimer(timer, "Sequential LUD with pivoting (GPU)", printDebugMessages);
        break;
    case 3:
        kernelName = "3";
        beginTimer(timer);

        endTimer(timer, "3", printDebugMessages);
        break;
    default:
        kernelName = "Unknown";
        break;
    }
    return kernelName;
}

int main(int argc, char *argv[])
{
    int KernelNumToPerform = atoi(argv[1]);
    int ADim = atoi(argv[2]); 

    size_t memorySize = ADim * ADim * sizeof(float);

    Timer timer = createTimer();

    beginTimer(timer);
    MatrixF A;
    float *device_A;
    
    A = createMatrixFloats(ADim, ADim);
    populateWithRandomFloats(A);

    hipMalloc((void **)&device_A, memorySize);
    hipMemcpy(device_A, A.data, memorySize, hipMemcpyHostToDevice);
    endTimer(timer, "initialize matrices on CPU and GPU", printDebugMessages);

    const char *kernelName = executeChosenKernel(KernelNumToPerform, device_A, ADim, timer);

    hipMemcpy(A.data, device_A, memorySize, hipMemcpyDeviceToHost);

    // Setup a CPU comparison matrix
    float** A_CPU_2D = MatrixF_to_twoDim(createMatrixFloats(ADim, ADim));
    printf("Setup 2d matrix, %f \n", A_CPU_2D);
    LUD_Sequential(A_CPU_2D, ADim);
    printf("CPU calculations done");
    MatrixF A_CPU_1D = twoDim_to_MatrixF(A_CPU_2D, ADim, ADim);
    printf("Converted to 1D matrix, %f", A_CPU_1D);

    // Validate result by comparing to CPU calculations
    bool valid = compareMatricesFloats(A_CPU_1D, A);
    if (valid)
    {
        printf("Matrix LUD results match!\n");
    }
    else
    {
        printf("Matrix LUD results do not match.\n");
        // Write the CPU Matrix to text file for analysis
        char fileNameCPU[100];
        sprintf(fileNameCPU, "resultsCPU.txt");

        printMatrixToFileFloats(fileNameCPU, A_CPU_1D);
    }

    char fileName[FILENAME_MAX_LENGTH];
    sprintf(fileName, "Test/LUD_%s_Runtime_Matrix_Size_%dx%d.csv", kernelName, ADim, ADim);
    printMatrixToFileFloats(fileName, A);

    hipFree(device_A);
    free(A.data);

    // Exit program
    return 0;
}