#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include "..\..\Matrix\matrixDoubles.cu"
#include "..\..\Timer\timer.cu"

__global__ void Sequential(double *M1, double *M2, double *M3, int M1Rows, int M1Cols, int M2Cols)
{
	for (int i = 0; i < M1Rows; i++)
	{
		for (int j = 0; j < M2Cols; j++)
		{
			double sum = 0.0;
			for (int k = 0; k < M1Cols; k++)
			{
				sum += M1[i * M1Cols + k] * M2[k * M2Cols + j];
			}
			M3[i * M2Cols + j] = sum;
		}
	}
}

__global__ void Parallel(double *M1, double *M2, double *M3, int M1Rows, int M1Cols, int M2Cols)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < M1Rows && col < M2Cols)
	{
		double sum = 0.0;

		for (int i = 0; i < M1Cols; i++)
		{
			sum += M1[row * M1Cols + i] * M2[i * M2Cols + col];
		}
		M3[row * M2Cols + col] = sum;
	}
}

__global__ void SharedMemoryAndTiling(double *M1, double *M2, double *M3, int M1Rows, int M1Cols, int M2Cols)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ double sharedMemory1[256];
	__shared__ double sharedMemory2[256];

	int sharedIndex = threadIdx.y * blockDim.x + threadIdx.x;

	double sum = 0.0;

	for (int i = 0; i < M1Cols; i += blockDim.x)
	{
		// Load M1 into shared memory
		if (row < M1Rows && (i + threadIdx.x) < M1Cols)
			sharedMemory1[sharedIndex] = M1[row * M1Cols + i + threadIdx.x];
		else
			sharedMemory1[sharedIndex] = 0;

		// Load M2 into shared memory
		if ((i + threadIdx.y) < M1Cols && col < M2Cols)
			sharedMemory2[sharedIndex] = M2[(i + threadIdx.y) * M2Cols + col];
		else
			sharedMemory2[sharedIndex] = 0;

		__syncthreads();

		// Tile multiplication
		int numIterations = (M1Cols - i > blockDim.x) ? blockDim.x : M1Cols - i;
		for (int j = 0; j < numIterations; j++)
		{
			sum += sharedMemory1[threadIdx.y * blockDim.x + j] * sharedMemory2[j * blockDim.x + threadIdx.x];
		}

		__syncthreads();
	}

	if (row < M1Rows && col < M2Cols)
	{
		M3[row * M2Cols + col] = sum;
	}
}