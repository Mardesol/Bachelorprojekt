#include "multiplicationFloatsKernels.cu"
#include "..\..\Matrix\matrixOperationsCPU.cu"
#include "..\..\Matrix\matrixCompatability.cu"

const bool printDebugMessages = false;

// Execute the chosen kernel
const char* executeChosenKernel(int KernelNumToPerform, float *device_M1, float *device_M2, float *device_M3, int M1Rows, int M1Cols, int M2Cols, Timer timer)
{
    dim3 blockDim(32, 32);
    dim3 gridDim((M1Cols + blockDim.x - 1) / blockDim.x, (M1Rows + blockDim.y - 1) / blockDim.y);
    const char *kernelName;

    switch (KernelNumToPerform)
    {
    case 1:
        kernelName = "Sequential";
        beginTimer(timer);
        Sequential<<<gridDim, blockDim>>>(device_M1, device_M2, device_M3, M1Rows, M1Cols, M2Cols);
        endTimer(timer, "Sequential matrix multiplication (GPU)", printDebugMessages);
        break;
    case 2:
        kernelName = "Parallel";
        beginTimer(timer);
        Parallel<<<gridDim, blockDim>>>(device_M1, device_M2, device_M3, M1Rows, M1Cols, M2Cols);
        endTimer(timer, "Parallel matrix multiplication (GPU)", printDebugMessages);
        break;
    case 3:
        kernelName = "SharedMemoryAndTiling";
        beginTimer(timer);
        SharedMemoryAndTiling<<<gridDim, blockDim>>>(device_M1, device_M2, device_M3, M1Rows, M1Cols, M2Cols);
        endTimer(timer, "SharedMemoryAndTiling matrix multiplication (GPU)", printDebugMessages);
        break;
    default:
        kernelName = "Unknown";
        break;
    }
    return kernelName;
}

int main(int argc, char *argv[])
{
    int KernelNumToPerform = atoi(argv[1]);
    int M1Rows = atoi(argv[2]);
    int M1Cols = atoi(argv[3]);
	int M2Rows = atoi(argv[4]);
    int M2Cols = atoi(argv[5]);
	int M3Rows = M1Rows;
    int M3Cols = M2Cols;

    size_t memorySize1 = M1Rows * M1Cols * sizeof(int);
	size_t memorySize2 = M2Rows * M2Cols * sizeof(int);
	size_t memorySize3 = M3Rows * M3Cols * sizeof(int);

    if (!isCompatibleForMultiplication(M1Cols, M2Rows))
    {
        perror("Matrices must be compatible");
        return 1;
    }

    // Timer measure time spent on a process
    Timer timer = createTimer();

    beginTimer(timer);
    MatrixF M1, M2, M3;
    float *device_M1, *device_M2, *device_M3;
    initializeMatricesAndMemory(M1, M2, M3, M1Rows, M1Cols, M2Rows, M2Cols, M3Rows, M3Cols);
    allocateMemoryOnGPU(device_M1, device_M2, device_M3, memorySize1, memorySize2, memorySize3);
    copyMatricesToGPU(M1, M2, device_M1, device_M2, memorySize1, memorySize2);
    endTimer(timer, "initialize matrices on CPU and GPU", printDebugMessages);

    // Define block and grid dimensions for CUDA kernel
    dim3 blockDim(16, 16);

    if (M3Rows <= 16 && M3Cols <= 16)
    {
        blockDim = dim3(M3Cols, M3Rows); // Use matrix size for smaller matrices
    }

    dim3 gridDim((M3Cols + blockDim.x - 1) / blockDim.x, (M3Rows + blockDim.y - 1) / blockDim.y);

    const char* kernelName = executeChosenKernel(KernelNumToPerform, device_M1, device_M2, device_M3, M1Rows, M1Cols, M2Cols, timer);

    // Copy the result matrix from device to host
    hipMemcpy(M3.data, device_M3, memorySize3, hipMemcpyDeviceToHost);

    // Setup a CPU comparison matrix
    MatrixF MCPU = createMatrixFloats(M3Rows, M3Cols);
    multiplicationFloats(M1, M2, MCPU);

    // Validate result by comparing to CPU calculations
    bool valid = compareMatricesFloats(MCPU, M3);
    if (valid)
    {
        printf("Matrix multiplication results match!\n");
    }
    else
    {
        printf("Matrix multiplication results do not match.\n");
        // Write the CPU matrix to text file for analysis
        char fileNameCPU[100];
        sprintf(fileNameCPU, "resultsIntsCPU.txt");

        printMatrixToFileFloats(fileNameCPU, MCPU);
    }

    // Open a new file to write the result into
    char fileName[100];                                                                                           // Max length filename (Just needs to be long enough)
    sprintf(fileName, "Test/Multiplication_%s_Floats_Runtime_Matrix_Size_%dx%d.csv", kernelName, M3Rows, M3Cols); // Customize filename to reflect size of result matrix
    printMatrixToFileFloats(fileName, M3);

    freeMemory(device_M1, device_M2, device_M3, M1, M2, M3);

    // Exit program
    return 0;
}